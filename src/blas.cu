#include "hip/hip_runtime.h"
#include <blas.cuh>
#include <handle.cuh>

#include <hipblas.h>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

namespace my_tensor {
template <>
void matmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B<sup>T</sup>)(A<sup>T</sup>)
  // also C = AB
  CUBLAS_ERROR_CHECK(hipblasSgemm(handle->GetHandle(),  // handle
    HIPBLAS_OP_N,  // no transpose of A<sup>T</sup>
    HIPBLAS_OP_N,  // no transpose of B<sup>T</sup>
    n,  // row number of B<sup>T</sup> and row number of C<sup>T</sup>
    m,  // col number of A<sup>T</sup> and col number of C<sup>T</sup>
    k,  // col number of B<sup>T</sup> and row number of A<sup>T</sup>
    &alpha,  // alpha
    B,  // B pointer, in cublas will be B<sup>T</sup>
    n,  // leading dimension of B<sup>T</sup>
    A,  // A pointer, in cublas will be A<sup>T</sup>
    k,  // leading dimension of A<sup>T</sup>
    &beta,  // beta
    C,  // C pointer, in cublas will be C<sup>T</sup>
    n  // leading dimension of C<sup>T</sup>
  ));
}

template <>
void transpose_matmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B<sup>T</sup>)(A)
  // also C = (A<sup>T</sup>)B
  CUBLAS_ERROR_CHECK(hipblasSgemm(handle->GetHandle(),  // handle
    HIPBLAS_OP_N,  // no transpose of A<sup>T</sup>
    HIPBLAS_OP_T,  // no transpose of B<sup>T</sup>
    n,  // row number of B<sup>T</sup> and row number of C<sup>T</sup>
    m,  // col number of A and col number of C<sup>T</sup>
    k,  // col number of B<sup>T</sup> and row number of A
    &alpha,  // alpha
    B,  // B pointer, in cublas will be B<sup>T</sup>
    n,  // leading dimension of B<sup>T</sup>
    A,  // A pointer, in cublas will be A<sup>T</sup>
    m,  // leading dimension of A<sup>T</sup>
    &beta,  // beta
    C,  // C pointer, in cublas will be C<sup>T</sup>
    n  // leading dimension of C<sup>T</sup>
  ));
}

template <>
void matmul_transpose(const float *A, const float *B, float *C, const int m, const int k, const int n) {
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B)(A<sup>T</sup>)
  // also C = A(B<sup>T</sup>)
  CUBLAS_ERROR_CHECK(hipblasSgemm(handle->GetHandle(),  // handle
    HIPBLAS_OP_T,  // no transpose of A<sup>T</sup>
    HIPBLAS_OP_N,  // no transpose of B<sup>T</sup>
    n,  // row number of B and row number of C<sup>T</sup>
    m,  // col number of A<sup>T</sup> and col number of C<sup>T</sup>
    k,  // col number of B and row number of A<sup>T</sup>
    &alpha,  // alpha
    B,  // B pointer, in cublas will be B<sup>T</sup>
    k,  // leading dimension of B<sup>T</sup>
    A,  // A pointer, in cublas will be A<sup>T</sup>
    k,  // leading dimension of A<sup>T</sup>
    &beta,  // beta
    C,  // C pointer, in cublas will be C<sup>T</sup>
    n  // leading dimension of C<sup>T</sup>
  ));
}

template <>
void transpose_matmul_transpose(const float *A, const float *B, float *C, const int m, const int k, const int n) {
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B)(A)
  // also C = (A<sup>T</sup>)(B<sup>T</sup>)
  CUBLAS_ERROR_CHECK(hipblasSgemm(handle->GetHandle(),  // handle
    HIPBLAS_OP_T,  // no transpose of A<sup>T</sup>
    HIPBLAS_OP_T,  // no transpose of B<sup>T</sup>
    n,  // row number of B and row number of C<sup>T</sup>
    m,  // col number of A and col number of C<sup>T</sup>
    k,  // col number of B and row number of A
    &alpha,  // alpha
    B,  // B pointer, in cublas will be B<sup>T</sup>
    k,  // leading dimension of B<sup>T</sup>
    A,  // A pointer, in cublas will be A<sup>T</sup>
    m,  // leading dimension of A<sup>T</sup>
    &beta,  // beta
    C,  // C pointer, in cublas will be C<sup>T</sup>
    n  // leading dimension of C<sup>T</sup>
  ));
}

namespace {
__global__ void SetAllOnes(float *ones, int n) {
  CUDA_KERNEL_LOOP(i, n) {
    ones[i] = 1.0f;
  }
}
}  // namespace

template <>
void add_row_vector(float *mat, const float *vec, const int m, const int n) {
  float alpha = 1.0f;
  float beta = 1.0f;
  float *ones = nullptr;
  hipMalloc(&ones, n * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(n), kCudaThreadNum>>>(ones, n);
  CUBLAS_ERROR_CHECK(hipblasSgemm(handle->GetHandle(),
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    n,
    m,
    1,
    &alpha,
    ones,
    n,
    vec,
    1,
    &beta,
    mat,
    n));
  hipFree(ones);
}

template <>
void add_col_vector(float *mat, const float *vec, const int m, const int n) {
  float alpha = 1.0f;
  float beta = 1.0f;
  float *ones = nullptr;
  hipMalloc(&ones, m * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(m), kCudaThreadNum>>>(ones, m);
  CUBLAS_ERROR_CHECK(hipblasSgemm(handle->GetHandle(),
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    n,
    m,
    1,
    &alpha,
    vec,
    n,
    ones,
    1,
    &beta,
    mat,
    n));
  hipFree(ones);
}

template <>
float tensor_sum(const float *tensor, const int cnt) {
  return thrust::reduce(thrust::device_pointer_cast(tensor),
                        thrust::device_pointer_cast(tensor + cnt),
                        0.0f, thrust::plus<float>());
}

template <>
void row_sum(const float *mat, float *result, const int m, const int n) {
  float *ones = nullptr;
  hipMalloc(&ones, n * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(n), kCudaThreadNum>>>(ones, n);
  matmul(mat, ones, result, m, n, 1);
  hipFree(ones);
}

template <>
void col_sum(const float *mat, float *result, const int m, const int n) {
  float *ones = nullptr;
  hipMalloc(&ones, m * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(m), kCudaThreadNum>>>(ones, m);
  transpose_matmul(mat, ones, result, n, m, 1);
  hipFree(ones);
}

}  // namespace my_tensor
