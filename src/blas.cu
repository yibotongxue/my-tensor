#include "hip/hip_runtime.h"
// Copyright 2024 yibotongxue

#include <blas.cuh>
#include <handle.cuh>

#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>

namespace my_tensor {
extern HandlePtr handle;

#define DEFINE_ABC_VEC(broadcast)\
  int stride_A = (broadcast == 1) ? 0 : (m * k);\
  thrust::device_vector<const float *> A_vec(batch_count);\
  thrust::transform(thrust::counting_iterator<int>(0), \
                    thrust::counting_iterator<int>(batch_count), \
                    A_vec.begin(), \
                    [A, stride_A] __device__(int i) -> const float* {\
                        return A + i * stride_A;\
                    });\
  int stride_B = (broadcast == 2) ? 0 : (k * n);\
  thrust::device_vector<const float *> B_vec(batch_count);\
  thrust::transform(thrust::counting_iterator<int>(0), \
                    thrust::counting_iterator<int>(batch_count), \
                    B_vec.begin(), \
                    [B, stride_B] __device__(int i) -> const float* {\
                        return B + i * stride_B;\
                    });\
  thrust::device_vector<float *> C_vec(batch_count);\
  thrust::transform(thrust::counting_iterator<int>(0), \
                    thrust::counting_iterator<int>(batch_count), \
                    C_vec.begin(), \
                    [C, m, n] __device__(int i) -> float* {\
                        return C + i * m * n;\
                    });

template <>
void matmul(const float *A, const float *B, float *C,
    const int m, const int k, const int n,
    const int batch_count, const int broadcast) {
  float alpha = 1.0f;
  float beta = 0.0f;
  DEFINE_ABC_VEC(broadcast)
  // C<sup>T</sup> = (B<sup>T</sup>)(A<sup>T</sup>)
  // also C = AB
  CUBLAS_ERROR_CHECK(hipblasSgemmBatched(handle->GetHandle(),  // handle
    HIPBLAS_OP_N,  // no transpose of A<sup>T</sup>
    HIPBLAS_OP_N,  // no transpose of B<sup>T</sup>
    n,  // row number of B<sup>T</sup> and row number of C<sup>T</sup>
    m,  // col number of A<sup>T</sup> and col number of C<sup>T</sup>
    k,  // col number of B<sup>T</sup> and row number of A<sup>T</sup>
    &alpha,  // alpha
    RAW_PTR(B_vec),  // B pointer, in cublas will be B<sup>T</sup>
    n,  // leading dimension of B<sup>T</sup>
    RAW_PTR(A_vec),  // A pointer, in cublas will be A<sup>T</sup>
    k,  // leading dimension of A<sup>T</sup>
    &beta,  // beta
    RAW_PTR(C_vec),  // C pointer, in cublas will be C<sup>T</sup>
    n,  // leading dimension of C<sup>T</sup>
    batch_count));
}

template <>
void transpose_matmul(const float *A, const float *B,
    float *C, const int m, const int k, const int n,
    const int batch_count, const int broadcast) {
  DEFINE_ABC_VEC(broadcast)
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B<sup>T</sup>)(A)
  // also C = (A<sup>T</sup>)B
  CUBLAS_ERROR_CHECK(hipblasSgemmBatched(handle->GetHandle(),  // handle
    HIPBLAS_OP_N,  // no transpose of A<sup>T</sup>
    HIPBLAS_OP_T,  // no transpose of B<sup>T</sup>
    n,  // row number of B<sup>T</sup> and row number of C<sup>T</sup>
    m,  // col number of A and col number of C<sup>T</sup>
    k,  // col number of B<sup>T</sup> and row number of A
    &alpha,  // alpha
    RAW_PTR(B_vec),  // B pointer, in cublas will be B<sup>T</sup>
    n,  // leading dimension of B<sup>T</sup>
    RAW_PTR(A_vec),  // A pointer, in cublas will be A<sup>T</sup>
    m,  // leading dimension of A<sup>T</sup>
    &beta,  // beta
    RAW_PTR(C_vec),  // C pointer, in cublas will be C<sup>T</sup>
    n,  // leading dimension of C<sup>T</sup>
    batch_count));
}

template <>
void matmul_transpose(const float *A, const float *B,
    float *C, const int m, const int k, const int n,
    const int batch_count, const int broadcast) {
  DEFINE_ABC_VEC(broadcast)
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B)(A<sup>T</sup>)
  // also C = A(B<sup>T</sup>)
  CUBLAS_ERROR_CHECK(hipblasSgemmBatched(handle->GetHandle(),  // handle
    HIPBLAS_OP_T,  // no transpose of A<sup>T</sup>
    HIPBLAS_OP_N,  // no transpose of B<sup>T</sup>
    n,  // row number of B and row number of C<sup>T</sup>
    m,  // col number of A<sup>T</sup> and col number of C<sup>T</sup>
    k,  // col number of B and row number of A<sup>T</sup>
    &alpha,  // alpha
    RAW_PTR(B_vec),  // B pointer, in cublas will be B<sup>T</sup>
    k,  // leading dimension of B<sup>T</sup>
    RAW_PTR(A_vec),  // A pointer, in cublas will be A<sup>T</sup>
    k,  // leading dimension of A<sup>T</sup>
    &beta,  // beta
    RAW_PTR(C_vec),  // C pointer, in cublas will be C<sup>T</sup>
    n,  // leading dimension of C<sup>T</sup>
    batch_count));
}

template <>
void transpose_matmul_transpose(const float *A,
    const float *B, float *C, const int m, const int k,
    const int n, const int batch_count,
    const int broadcast) {
  DEFINE_ABC_VEC(broadcast)
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B)(A)
  // also C = (A<sup>T</sup>)(B<sup>T</sup>)
  CUBLAS_ERROR_CHECK(hipblasSgemmBatched(handle->GetHandle(),  // handle
    HIPBLAS_OP_T,  // no transpose of A<sup>T</sup>
    HIPBLAS_OP_T,  // no transpose of B<sup>T</sup>
    n,  // row number of B and row number of C<sup>T</sup>
    m,  // col number of A and col number of C<sup>T</sup>
    k,  // col number of B and row number of A
    &alpha,  // alpha
    RAW_PTR(B_vec),  // B pointer, in cublas will be B<sup>T</sup>
    k,  // leading dimension of B<sup>T</sup>
    RAW_PTR(A_vec),  // A pointer, in cublas will be A<sup>T</sup>
    m,  // leading dimension of A<sup>T</sup>
    &beta,  // beta
    RAW_PTR(C_vec),  // C pointer, in cublas will be C<sup>T</sup>
    n,  // leading dimension of C<sup>T</sup>
    batch_count));
}

namespace {
__global__ void SetAllOnes(float *ones, int n) {
  CUDA_KERNEL_LOOP(i, n) {
    ones[i] = 1.0f;
  }
}
}  // namespace

template <>
void add_row_vector(float *mat, const float *vec, const int m, const int n) {
  float alpha = 1.0f;
  float *ones = nullptr;
  hipMalloc(&ones, sizeof(float));
  SetAllOnes<<<CudaGetBlocks(n), kCudaThreadNum>>>(ones, n);
  CUBLAS_ERROR_CHECK(hipblasSger(handle->GetHandle(),
    n,
    m,
    &alpha,
    ones,
    1,
    vec,
    1,
    mat,
    n));
  hipFree(ones);
}

template <>
void add_col_vector(float *mat, const float *vec, const int m, const int n) {
  float alpha = 1.0f;
  float *ones = nullptr;
  hipMalloc(&ones, m * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(m), kCudaThreadNum>>>(ones, m);
  CUBLAS_ERROR_CHECK(hipblasSger(handle->GetHandle(),
    n,
    m,
    &alpha,
    vec,
    1,
    ones,
    1,
    mat,
    n));
  hipFree(ones);
}

template <>
float tensor_sum(const float *tensor, const int cnt) {
  return thrust::reduce(thrust::device_pointer_cast(tensor),
                        thrust::device_pointer_cast(tensor + cnt),
                        0.0f, thrust::plus<float>());
}

template <>
void row_sum(const float *mat, float *result, const int m, const int n) {
  float alpha = 1.0f;
  float beta = 0.0f;
  float *ones = nullptr;
  hipMalloc(&ones, n * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(n), kCudaThreadNum>>>(ones, n);
  CUBLAS_ERROR_CHECK(hipblasSgemv(handle->GetHandle(),
    HIPBLAS_OP_T,
    n,
    m,
    &alpha,
    mat,
    n,
    ones,
    1,
    &beta,
    result,
    1));
}

template <>
void col_sum(const float *mat, float *result, const int m, const int n) {
  float alpha = 1.0f;
  float beta = 0.0f;
  float *ones = nullptr;
  hipMalloc(&ones, m * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(m), kCudaThreadNum>>>(ones, m);
  CUBLAS_ERROR_CHECK(hipblasSgemv(handle->GetHandle(),
    HIPBLAS_OP_N,
    n,
    m,
    &alpha,
    mat,
    n,
    ones,
    1,
    &beta,
    result,
    1));
  hipFree(ones);
}

}  // namespace my_tensor
