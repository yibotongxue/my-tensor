#include "hip/hip_runtime.h"
// Copyright 2024 yibotongxue

#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>

#include "blas.hpp"
#include "common.hpp"

namespace my_tensor {

#define DEFINE_ABC_VEC(broadcast)                                      \
  int stride_A = (broadcast == 1) ? 0 : (m * k);                       \
  thrust::device_vector<const float *> A_vec(batch_count);             \
  thrust::transform(thrust::counting_iterator<int>(0),                 \
                    thrust::counting_iterator<int>(batch_count),       \
                    A_vec.begin(),                                     \
                    [A, stride_A] __device__(int i) -> const float * { \
                      return A + i * stride_A;                         \
                    });                                                \
  int stride_B = (broadcast == 2) ? 0 : (k * n);                       \
  thrust::device_vector<const float *> B_vec(batch_count);             \
  thrust::transform(thrust::counting_iterator<int>(0),                 \
                    thrust::counting_iterator<int>(batch_count),       \
                    B_vec.begin(),                                     \
                    [B, stride_B] __device__(int i) -> const float * { \
                      return B + i * stride_B;                         \
                    });                                                \
  thrust::device_vector<float *> C_vec(batch_count);                   \
  thrust::transform(                                                   \
      thrust::counting_iterator<int>(0),                               \
      thrust::counting_iterator<int>(batch_count), C_vec.begin(),      \
      [C, m, n] __device__(int i) -> float * { return C + i * m * n; });

template <>
void matmul_gpu(const float *A, const float *B, float *C, const int m,
                const int k, const int n, const int batch_count,
                const int broadcast) {
  float alpha = 1.0f;
  float beta = 0.0f;
  DEFINE_ABC_VEC(broadcast)
  // C<sup>T</sup> = (B<sup>T</sup>)(A<sup>T</sup>)
  // also C = AB
  CUBLAS_CHECK(hipblasSgemmBatched(
      MyTensorContext::cublas_handle(),  // handle
      HIPBLAS_OP_N,                       // no transpose of A<sup>T</sup>
      HIPBLAS_OP_N,                       // no transpose of B<sup>T</sup>
      n,       // row number of B<sup>T</sup> and row number of C<sup>T</sup>
      m,       // col number of A<sup>T</sup> and col number of C<sup>T</sup>
      k,       // col number of B<sup>T</sup> and row number of A<sup>T</sup>
      &alpha,  // alpha
      RAW_PTR(B_vec),  // B pointer, in cublas will be B<sup>T</sup>
      n,               // leading dimension of B<sup>T</sup>
      RAW_PTR(A_vec),  // A pointer, in cublas will be A<sup>T</sup>
      k,               // leading dimension of A<sup>T</sup>
      &beta,           // beta
      RAW_PTR(C_vec),  // C pointer, in cublas will be C<sup>T</sup>
      n,               // leading dimension of C<sup>T</sup>
      batch_count));
}

template <>
void transpose_matmul_gpu(const float *A, const float *B, float *C, const int m,
                          const int k, const int n, const int batch_count,
                          const int broadcast) {
  DEFINE_ABC_VEC(broadcast)
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B<sup>T</sup>)(A)
  // also C = (A<sup>T</sup>)B
  CUBLAS_CHECK(hipblasSgemmBatched(
      MyTensorContext::cublas_handle(),  // handle
      HIPBLAS_OP_N,                       // no transpose of A<sup>T</sup>
      HIPBLAS_OP_T,                       // no transpose of B<sup>T</sup>
      n,       // row number of B<sup>T</sup> and row number of C<sup>T</sup>
      m,       // col number of A and col number of C<sup>T</sup>
      k,       // col number of B<sup>T</sup> and row number of A
      &alpha,  // alpha
      RAW_PTR(B_vec),  // B pointer, in cublas will be B<sup>T</sup>
      n,               // leading dimension of B<sup>T</sup>
      RAW_PTR(A_vec),  // A pointer, in cublas will be A<sup>T</sup>
      m,               // leading dimension of A<sup>T</sup>
      &beta,           // beta
      RAW_PTR(C_vec),  // C pointer, in cublas will be C<sup>T</sup>
      n,               // leading dimension of C<sup>T</sup>
      batch_count));
}

template <>
void matmul_transpose_gpu(const float *A, const float *B, float *C, const int m,
                          const int k, const int n, const int batch_count,
                          const int broadcast) {
  DEFINE_ABC_VEC(broadcast)
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B)(A<sup>T</sup>)
  // also C = A(B<sup>T</sup>)
  CUBLAS_CHECK(hipblasSgemmBatched(
      MyTensorContext::cublas_handle(),  // handle
      HIPBLAS_OP_T,                       // no transpose of A<sup>T</sup>
      HIPBLAS_OP_N,                       // no transpose of B<sup>T</sup>
      n,       // row number of B and row number of C<sup>T</sup>
      m,       // col number of A<sup>T</sup> and col number of C<sup>T</sup>
      k,       // col number of B and row number of A<sup>T</sup>
      &alpha,  // alpha
      RAW_PTR(B_vec),  // B pointer, in cublas will be B<sup>T</sup>
      k,               // leading dimension of B<sup>T</sup>
      RAW_PTR(A_vec),  // A pointer, in cublas will be A<sup>T</sup>
      k,               // leading dimension of A<sup>T</sup>
      &beta,           // beta
      RAW_PTR(C_vec),  // C pointer, in cublas will be C<sup>T</sup>
      n,               // leading dimension of C<sup>T</sup>
      batch_count));
}

template <>
void transpose_matmul_transpose_gpu(const float *A, const float *B, float *C,
                                    const int m, const int k, const int n,
                                    const int batch_count,
                                    const int broadcast) {
  DEFINE_ABC_VEC(broadcast)
  float alpha = 1.0f;
  float beta = 0.0f;
  // C<sup>T</sup> = (B)(A)
  // also C = (A<sup>T</sup>)(B<sup>T</sup>)
  CUBLAS_CHECK(hipblasSgemmBatched(
      MyTensorContext::cublas_handle(),  // handle
      HIPBLAS_OP_T,                       // no transpose of A<sup>T</sup>
      HIPBLAS_OP_T,                       // no transpose of B<sup>T</sup>
      n,               // row number of B and row number of C<sup>T</sup>
      m,               // col number of A and col number of C<sup>T</sup>
      k,               // col number of B and row number of A
      &alpha,          // alpha
      RAW_PTR(B_vec),  // B pointer, in cublas will be B<sup>T</sup>
      k,               // leading dimension of B<sup>T</sup>
      RAW_PTR(A_vec),  // A pointer, in cublas will be A<sup>T</sup>
      m,               // leading dimension of A<sup>T</sup>
      &beta,           // beta
      RAW_PTR(C_vec),  // C pointer, in cublas will be C<sup>T</sup>
      n,               // leading dimension of C<sup>T</sup>
      batch_count));
}

namespace {
__global__ void SetAllOnes(float *ones, int n) {
  CUDA_KERNEL_LOOP(i, n) { ones[i] = 1.0f; }
}

__global__ void RepeatVec(const float *vec, float *result, const int m,
                          const int batch_count) {
  CUDA_KERNEL_LOOP(i, m * batch_count) { result[i] = vec[i % m]; }
}
}  // namespace

template <>
void add_row_vector_gpu(float *mat, const float *vec, const int m, const int n,
                        const int batch_count) {
  float alpha = 1.0f;
  float *ones = nullptr;
  hipMalloc(&ones, n * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(n), kCudaThreadNum>>>(ones, n);
  float *repeat_vec = nullptr;
  hipMalloc(&repeat_vec, m * batch_count * sizeof(float));
  RepeatVec<<<CudaGetBlocks(m * batch_count), kCudaThreadNum>>>(vec, repeat_vec,
                                                                m, batch_count);
  CUBLAS_CHECK(hipblasSger(MyTensorContext::cublas_handle(), n, m * batch_count,
                          &alpha, ones, 1, repeat_vec, 1, mat, n));
  hipFree(ones);
  hipFree(repeat_vec);
}

template <>
void add_col_vector_gpu(float *mat, const float *vec, const int m, const int n,
                        const int batch_count) {
  float alpha = 1.0f;
  float *ones = nullptr;
  hipMalloc(&ones, m * batch_count * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(m * batch_count), kCudaThreadNum>>>(
      ones, m * batch_count);
  CUBLAS_CHECK(hipblasSger(MyTensorContext::cublas_handle(), n, m * batch_count,
                          &alpha, vec, 1, ones, 1, mat, n));
  hipFree(ones);
}

template <>
float tensor_sum_gpu(const float *tensor, const int cnt) {
  return thrust::reduce(thrust::device_pointer_cast(tensor),
                        thrust::device_pointer_cast(tensor + cnt), 0.0f,
                        thrust::plus<float>());
}

template <>
void row_sum_gpu(const float *mat, float *result, const int m, const int n,
                 const int batch_count) {
  float alpha = 1.0f;
  float beta = 0.0f;
  float *ones = nullptr;
  hipMalloc(&ones, n * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(n), kCudaThreadNum>>>(ones, n);
  CUBLAS_CHECK(hipblasSgemv(MyTensorContext::cublas_handle(), HIPBLAS_OP_T, n,
                           m * batch_count, &alpha, mat, n, ones, 1, &beta,
                           result, 1));
  hipFree(ones);
}

template <>
void col_sum_gpu(const float *mat, float *result, const int m, const int n,
                 const int batch_count) {
  float alpha = 1.0f;
  float beta = 0.0f;
  float *ones = nullptr;
  hipMalloc(&ones, m * sizeof(float));
  SetAllOnes<<<CudaGetBlocks(m), kCudaThreadNum>>>(ones, m * batch_count);
  thrust::device_vector<const float *> mat_vec(batch_count);
  int mat_stride = m * n;
  thrust::transform(thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(batch_count),
                    mat_vec.begin(),
                    [mat, mat_stride] __device__(int i) -> const float * {
                      return mat + i * mat_stride;
                    });
  thrust::device_vector<const float *> ones_vec(batch_count);
  thrust::fill(ones_vec.begin(), ones_vec.end(), ones);
  thrust::device_vector<float *> result_vec(batch_count);
  thrust::transform(
      thrust::counting_iterator<int>(0),
      thrust::counting_iterator<int>(batch_count), result_vec.begin(),
      [result, n] __device__(int i) -> float * { return result + i * n; });
  CUBLAS_CHECK(hipblasSgemvBatched(MyTensorContext::cublas_handle(), HIPBLAS_OP_N,
                                  n, m, &alpha, RAW_PTR(mat_vec), n,
                                  RAW_PTR(ones_vec), 1, &beta,
                                  RAW_PTR(result_vec), 1, batch_count));
  hipFree(ones);
}

template <>
void add_two_vec_gpu(float *lhs, const float *rhs, const float k, const int n) {
  CUBLAS_CHECK(
      hipblasSaxpy(MyTensorContext::cublas_handle(), n, &k, rhs, 1, lhs, 1));
}

#undef DEFINE_ABC_VEC

}  // namespace my_tensor
