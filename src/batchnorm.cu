#include "hip/hip_runtime.h"
// Copyright 2025 yibotongxue

#include <memory>
#include <vector>

#include "batchnorm.hpp"
#include "blas.hpp"
#include "error.hpp"
#include "memory-util.hpp"

namespace my_tensor {

namespace {
template <typename T>
__global__ void ComputeMeanAndVariance(const T* const mean_data,
                                       T* const standarded_cache,
                                       const T* const bottom_data,
                                       T* const temp_cache2,
                                       const int batch_size, const int channels,
                                       const int spatial_size) {
  extern __shared__ T shared_vec[];
  int thread_id = threadIdx.x;
  if (thread_id < channels) {
    shared_vec[thread_id] = mean_data[thread_id];
  }
  __syncthreads();
  CUDA_KERNEL_LOOP(idx, batch_size * channels * spatial_size) {
    standarded_cache[idx] =
        bottom_data[idx] - shared_vec[(idx / spatial_size) % channels];
    temp_cache2[idx] = standarded_cache[idx] * standarded_cache[idx];
  }
}

template <typename T>
__global__ void ComputeCache(T* const variance_data, const T* const mean_data,
                             T* const mean_cache, T* const variance_cache,
                             const T move_scale_factor, const int channels,
                             const float n) {
  CUDA_KERNEL_LOOP(idx, channels) {
    variance_data[idx] = std::sqrt(variance_data[idx] / n);
    mean_cache[idx] = move_scale_factor * mean_cache[idx] +
                      (1.0 - move_scale_factor) * mean_data[idx];
    variance_cache[idx] = move_scale_factor * variance_cache[idx] +
                          (1.0 - move_scale_factor) * variance_data[idx];
  }
}

template <typename T>
__global__ void ComputeTopData(T* const top_data,
                               const T* const standared_cache,
                               const T* const gama_data,
                               const T* const beta_data, const int batch_size,
                               const int channels, const int spatial_size) {
  extern __shared__ T shared_vec[];
  int thread_id = threadIdx.x;
  if (thread_id < channels) {
    shared_vec[thread_id] = gama_data[thread_id];
  } else if (thread_id < 2 * channels) {
    shared_vec[thread_id] = beta_data[thread_id - channels];
  }
  __syncthreads();
  CUDA_KERNEL_LOOP(idx, batch_size * channels * spatial_size) {
    top_data[idx] =
        shared_vec[(idx / spatial_size) % channels] * standared_cache[idx] +
        shared_vec[(idx / spatial_size) % channels + channels];
  }
}
}  // namespace

template <typename T>
void BatchNorm<T>::ForwardGPU(const std::vector<TensorPtr<T>>& bottom,
                              const std::vector<TensorPtr<T>>& top) {
  const T* bottom_data = bottom[0]->GetGPUDataPtr();
  T* top_data = top[0]->GetGPUDataPtr();
  const T* gama_data = gama_->GetGPUDataPtr();
  const T* beta_data = beta_->GetGPUDataPtr();
  T* mean_data = mean_->GetGPUDataPtr();
  T* variance_data = sqrt_variance_->GetGPUDataPtr();
  if (this->is_train_) {
    Fill_GPU<T>(mean_data, channels_, 0);
    Fill_GPU<T>(variance_data, channels_, 0);
    row_sum_gpu(bottom_data, temp_cache1_->GetGPUDataPtr(), channels_,
                spatial_size_, batch_size_, all_ones_->GetGPUDataPtr());
    col_sum_gpu(temp_cache1_->GetGPUDataPtr(), mean_data, batch_size_,
                channels_, 1, all_ones_->GetGPUDataPtr());
    scale_gpu(
        mean_data, channels_,
        static_cast<T>(1.0) / static_cast<T>(batch_size_ * spatial_size_));
    ComputeMeanAndVariance<<<CudaGetBlocks(batch_size_ * channels_ *
                                           spatial_size_),
                             kCudaThreadNum, channels_ * sizeof(T)>>>(
        mean_data, standarded_cache_->GetGPUDataPtr(), bottom_data,
        temp_cache2_->GetGPUDataPtr(), batch_size_, channels_, spatial_size_);
    row_sum_gpu(temp_cache2_->GetGPUDataPtr(), temp_cache1_->GetGPUDataPtr(),
                channels_, spatial_size_, batch_size_,
                all_ones_->GetGPUDataPtr());
    col_sum_gpu(temp_cache1_->GetGPUDataPtr(), variance_data, batch_size_,
                channels_, 1, all_ones_->GetGPUDataPtr());
    ComputeCache<<<CudaGetBlocks(channels_), kCudaThreadNum>>>(
        variance_data, mean_data, mean_cache_->GetGPUDataPtr(),
        sqrt_variance_cache_->GetGPUDataPtr(), move_scale_factor_, channels_,
        batch_size_ * spatial_size_);
    divide_row_vector_gpu<T>(standarded_cache_->GetGPUDataPtr(), variance_data,
                             channels_, spatial_size_, batch_size_,
                             static_cast<T>(1e-5));
  } else {
    mean_data = mean_cache_->GetGPUDataPtr();
    variance_data = sqrt_variance_cache_->GetGPUDataPtr();
    MyMemcpyGPU2GPU(standarded_cache_->GetGPUDataPtr(), bottom_data,
                    bottom[0]->GetSize() * sizeof(T));
    add_row_vector_gpu<T>(standarded_cache_->GetGPUDataPtr(), mean_data,
                          channels_, spatial_size_, batch_size_, -1);
    divide_row_vector_gpu<T>(standarded_cache_->GetGPUDataPtr(), variance_data,
                             channels_, spatial_size_, batch_size_,
                             static_cast<T>(1e-5));
  }
  ComputeTopData<<<CudaGetBlocks(batch_size_ * channels_ * spatial_size_),
                   kCudaThreadNum, 2 * channels_ * sizeof(T)>>>(
      top_data, standarded_cache_->GetGPUDataPtr(), gama_data, beta_data,
      batch_size_, channels_, spatial_size_);
}

namespace {
template <typename T>
__global__ void MutiplyRowVectorAndAssign(
    T* const target, const T* const source, const T* const vec,
    const int channels, const int spatial_size, const int batch_size) {
  extern __shared__ T shared_vec[];
  int thread_id = threadIdx.x;
  if (thread_id < channels) {
    shared_vec[thread_id] = vec[thread_id];
  }
  __syncthreads();
  CUDA_KERNEL_LOOP(idx, batch_size * channels * spatial_size) {
    target[idx] = source[idx] * shared_vec[(idx / spatial_size) % channels];
  }
}

template <typename T>
__global__ void ScaleBottomAndUpdateTempCache2(
    T* const bottom_diff, T* const temp_cache2, const T* const temp_cache,
    const T* const standarded_cache, const int batch_size, const int channels,
    const int spatial_size) {
  extern __shared__ T shared_vec[];
  int thread_id = threadIdx.x;
  if (thread_id < channels) {
    shared_vec[thread_id] = temp_cache[thread_id];
  }
  __syncthreads();
  int n = batch_size * spatial_size;
  CUDA_KERNEL_LOOP(idx, batch_size * channels * spatial_size) {
    bottom_diff[idx] =
        temp_cache2[idx] * n - shared_vec[(idx / spatial_size) % channels];
    temp_cache2[idx] *= standarded_cache[idx];
  }
}

template <typename T>
__global__ void ComputeBottomDiff(const T* const standarded_cache,
                                  const T* const temp_cache,
                                  T* const bottom_diff,
                                  const T* const variance_data,
                                  const int batch_size, const int channels,
                                  const int spatial_size) {
  extern __shared__ T shared_vec[];
  int thread_id = threadIdx.x;
  if (thread_id < channels) {
    shared_vec[thread_id] = temp_cache[thread_id];
  } else if (thread_id < 2 * channels) {
    shared_vec[thread_id] = variance_data[thread_id - channels];
  }
  __syncthreads();
  float n = batch_size * spatial_size;
  CUDA_KERNEL_LOOP(idx, batch_size * channels * spatial_size) {
    bottom_diff[idx] -=
        standarded_cache[idx] * shared_vec[(idx / spatial_size) % channels];
    bottom_diff[idx] /=
        (shared_vec[(idx / spatial_size) % channels + channels] + 1e-5) * n;
  }
}
}  // namespace

template <typename T>
void BatchNorm<T>::BackwardGPU(const std::vector<TensorPtr<T>>& top,
                               const std::vector<TensorPtr<T>>& bottom) {
  const T* top_diff = top[0]->GetGPUDiffPtr();
  const T* top_data = top[0]->GetGPUDataPtr();
  const T* bottom_data = bottom[0]->GetGPUDataPtr();
  T* bottom_diff = bottom[0]->GetGPUDiffPtr();
  const T* gama_data = gama_->GetGPUDataPtr();
  T* gama_diff = gama_->GetGPUDiffPtr();
  T* beta_diff = beta_->GetGPUDiffPtr();
  const T* mean_data = mean_->GetGPUDataPtr();
  const T* variance_data = sqrt_variance_->GetGPUDataPtr();
  row_sum_gpu(top_diff, temp_cache1_->GetGPUDataPtr(), channels_, spatial_size_,
              batch_size_, all_ones_->GetGPUDataPtr());
  col_sum_gpu(temp_cache1_->GetGPUDataPtr(), beta_diff, batch_size_, channels_,
              1, all_ones_->GetGPUDataPtr());
  multiply_two_vec_gpu<T>(top_diff, standarded_cache_->GetGPUDataPtr(),
                          temp_cache2_->GetGPUDataPtr(),
                          batch_size_ * channels_ * spatial_size_);
  row_sum_gpu(temp_cache2_->GetGPUDataPtr(), temp_cache1_->GetGPUDataPtr(),
              channels_, spatial_size_, batch_size_,
              all_ones_->GetGPUDataPtr());
  col_sum_gpu(temp_cache1_->GetGPUDataPtr(), gama_diff, batch_size_, channels_,
              1, all_ones_->GetGPUDataPtr());
  MutiplyRowVectorAndAssign<<<CudaGetBlocks(channels_ * spatial_size_ *
                                            batch_size_),
                              kCudaThreadNum, channels_ * sizeof(T)>>>(
      temp_cache2_->GetGPUDataPtr(), top_diff, gama_data, channels_,
      spatial_size_, batch_size_);
  row_sum_gpu(temp_cache2_->GetGPUDataPtr(), temp_cache1_->GetGPUDataPtr(),
              channels_, spatial_size_, batch_size_,
              all_ones_->GetGPUDataPtr());
  col_sum_gpu(temp_cache1_->GetGPUDataPtr(), temp_cache_->GetGPUDataPtr(),
              batch_size_, channels_, 1, all_ones_->GetGPUDataPtr());
  ScaleBottomAndUpdateTempCache2<<<CudaGetBlocks(batch_size_ * channels_ *
                                                 spatial_size_),
                                   kCudaThreadNum, channels_ * sizeof(T)>>>(
      bottom_diff, temp_cache2_->GetGPUDataPtr(), temp_cache_->GetGPUDataPtr(),
      standarded_cache_->GetGPUDataPtr(), batch_size_, channels_,
      spatial_size_);
  row_sum_gpu(temp_cache2_->GetGPUDataPtr(), temp_cache1_->GetGPUDataPtr(),
              channels_, spatial_size_, batch_size_,
              all_ones_->GetGPUDataPtr());
  col_sum_gpu(temp_cache1_->GetGPUDataPtr(), temp_cache_->GetGPUDataPtr(),
              batch_size_, channels_, 1, all_ones_->GetGPUDataPtr());
  ComputeBottomDiff<<<CudaGetBlocks(batch_size_ * channels_ * spatial_size_),
                      kCudaThreadNum, 2 * channels_ * sizeof(T)>>>(
      standarded_cache_->GetGPUDataPtr(), temp_cache_->GetGPUDataPtr(),
      bottom_diff, variance_data, batch_size_, channels_, spatial_size_);
}

template class BatchNorm<float>;

}  // namespace my_tensor
