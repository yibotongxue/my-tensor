#include "hip/hip_runtime.h"
// Copyright 2024 yibotongxue

#include <thrust/fill.h>
#include <thrust/scatter.h>

#include <iostream>
#include <memory>
#include <vector>

#include "error.h"
#include "layer-parameter.h"
#include "pooling.cuh"
#include "tensor.cuh"
#include "utils.cuh"

namespace my_tensor {

template <typename T>
void Pooling<T>::CheckTensorCount(const std::vector<TensorPtr<T>>& bottom,
                                  const std::vector<TensorPtr<T>>& top) const {
  if (bottom.size() != 1) {
    throw PoolingError(
        "The bottom of convolution layer should have one tensor.");
  }
  if (top.size() != 1) {
    throw PoolingError("The top of convolution layer should have one tensor.");
  }
}

template <typename T>
void Pooling<T>::Reshape(const std::vector<TensorPtr<T>>& bottom,
                         const std::vector<TensorPtr<T>>& top) const {
  int expect_size =
      batch_size_ * input_channels_ * output_height_ * output_width_;
  if (top[0]->GetSize() != expect_size) {
    throw PoolingError("The top size not match pooling layer.");
  }
  top[0]->Reshape(
      {batch_size_, input_channels_, output_height_, output_width_});
}

template <typename T>
void Pooling<T>::LayerSetUp(const std::vector<TensorPtr<T>>& bottom,
                            const std::vector<TensorPtr<T>>& top) {
  std::shared_ptr<PoolingParameter> param =
      std::dynamic_pointer_cast<PoolingParameter>(this->layer_param_);
  assert(param.get() != nullptr);
  input_channels_ = param->input_channels_;
  kernel_h_ = param->kernel_h_;
  kernel_w_ = param->kernel_w_;
  stride_h_ = param->stride_h_;
  stride_w_ = param->stride_w_;
  if (bottom[0]->GetShape().size() != 4) {
    throw PoolingError(
        "The input of pooling layer should be 4 dimension tensor.");
  }
  batch_size_ = bottom[0]->GetShape()[0];
  if (bottom[0]->GetShape()[1] != input_channels_) {
    throw PoolingError("The input channels not match.");
  }
  input_height_ = bottom[0]->GetShape()[2];
  input_width_ = bottom[0]->GetShape()[3];
  output_height_ = (input_height_ - kernel_h_) / stride_h_ + 1;
  output_width_ = (input_width_ - kernel_w_) / stride_w_ + 1;
  const std::vector<int> mask_shape = {batch_size_, input_channels_,
                                       output_height_, output_width_};
  mask_.reset();
  mask_ = std::make_shared<Tensor<int>>(mask_shape);
}

template <typename T>
void Pooling<T>::ForwardCPU(const std::vector<TensorPtr<T>>& bottom,
                            const std::vector<TensorPtr<T>>& top) {
  CheckShape(bottom[0], top[0]);
  const auto& bottom_data = bottom[0]->GetCPUData();
  auto& top_data = top[0]->GetCPUData();
  auto& mask_data = mask_->GetCPUData();
  int input_im_size = input_height_ * input_width_;
  int output_im_size = output_height_ * output_width_;
  for (int t = 0; t < batch_size_ * input_channels_; t++) {
    for (int i = 0; i < output_height_; i++) {
      for (int j = 0; j < output_width_; j++) {
        int h_start = i * stride_h_;
        int w_start = j * stride_w_;
        T val = static_cast<T>(-__FLT_MAX__);
        int mask_idx = -1;
        for (int x = 0; x < kernel_h_; x++) {
          for (int y = 0; y < kernel_w_; y++) {
            int temp_idx =
                t * input_im_size + (h_start + x) * input_width_ + w_start + y;
            T temp = bottom_data[temp_idx];
            if (temp > val) {
              val = temp;
              mask_idx = temp_idx;
            }
          }
        }
        top_data[t * output_im_size + i * output_width_ + j] = val;
        mask_data[t * output_im_size + i * output_width_ + j] = mask_idx;
      }
    }
  }
}

namespace {
template <typename T>
__global__ void PoolingKernel(const int nthreads, const T* const bottom_data,
                              const int n, const int input_w,
                              const int input_size, const int output_w,
                              const int output_size, const int kernel_h,
                              const int kernel_w, const int stride_h,
                              const int stride_w, T* top_data, int* mask_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int t = index / output_size;
    int h_start = (index % output_size) / output_w * stride_h;
    int w_start = (index % output_w) * stride_w;
    T val = static_cast<T>(-__FLT_MAX__);
    int idx = -1;
    int row_idx = t * input_size + h_start * input_w + w_start;
    for (int i = 0; i < kernel_h; i++) {
      int col_idx = row_idx;
      for (int j = 0; j < kernel_w; j++) {
        if (val < bottom_data[col_idx]) {
          val = bottom_data[col_idx];
          idx = col_idx;
        }
        col_idx += 1;
      }
      row_idx += input_w;
    }
    top_data[index] = val;
    mask_data[index] = idx;
  }
}
}  // namespace

template <typename T>
void Pooling<T>::ForwardGPU(const std::vector<TensorPtr<T>>& bottom,
                            const std::vector<TensorPtr<T>>& top) {
  CheckShape(bottom[0], top[0]);
  int input_size = input_height_ * input_width_;
  int output_size = output_height_ * output_width_;
  int n = batch_size_ * input_channels_;
  int nthreads = n * output_size;
  PoolingKernel<<<CudaGetBlocks(nthreads), kCudaThreadNum>>>(
      nthreads, bottom[0]->GetGPUDataPtr(), n, input_width_, input_size,
      output_width_, output_size, kernel_h_, kernel_w_, stride_h_, stride_w_,
      top[0]->GetGPUDataPtr(), mask_->GetGPUDataPtr());
}

template <typename T>
void Pooling<T>::BackwardCPU(const std::vector<TensorPtr<T>>& top,
                             const std::vector<TensorPtr<T>>& bottom) {
  CheckShape(bottom[0], top[0]);
  const auto& top_diff = top[0]->GetCPUDiff();
  const auto& mask_data = mask_->GetCPUData();
  auto& bottom_diff = bottom[0]->GetCPUDiff();
  thrust::fill(bottom_diff.begin(), bottom_diff.end(), 0);
  thrust::scatter(top_diff.begin(), top_diff.end(), mask_data.begin(),
                  bottom_diff.begin());
}

template <typename T>
void Pooling<T>::BackwardGPU(const std::vector<TensorPtr<T>>& top,
                             const std::vector<TensorPtr<T>>& bottom) {
  CheckShape(bottom[0], top[0]);
  const auto& top_diff = top[0]->GetGPUDiff();
  const auto& mask_data = mask_->GetGPUData();
  auto& bottom_diff = bottom[0]->GetGPUDiff();
  thrust::fill(bottom_diff.begin(), bottom_diff.end(), 0);
  thrust::scatter(top_diff.begin(), top_diff.end(), mask_data.begin(),
                  bottom_diff.begin());
}

template <typename T>
void Pooling<T>::CheckShape(const TensorPtr<T> bottom,
                            const TensorPtr<T> top) const {
#ifdef DEBUG
  const auto& bottom_shape = bottom->GetShape();
  const auto& top_shape = top->GetShape();
  if (bottom_shape.size() != 4) {
    throw PoolingError(
        "The input of pooling layer should be 4 dimension tensor.");
  }
  if (top_shape.size() != 4) {
    throw PoolingError(
        "The output of pooling layer should be 4 dimension tensor.");
  }
  if (bottom_shape[0] != top_shape[0]) {
    throw PoolingError(
        "The input and output of pooling layer should have the same batch "
        "size.");
  }
  if (bottom_shape[1] != top_shape[1]) {
    throw PoolingError(
        "The input and output of pooling layer should have the same channels.");
  }
  if (bottom_shape[2] != input_height_ || bottom_shape[3] != input_width_) {
    throw PoolingError("The input shape not match the pooling layer.");
  }
  if (top_shape[2] != output_height_ || top_shape[3] != output_width_) {
    throw PoolingError("The output shape not match the pooling layer.");
  }
#endif  // DEBUG
}

template class Pooling<>;

}  // namespace my_tensor
