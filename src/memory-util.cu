// Copyright 2024 yibotongxue

#include "memory-util.hpp"
#include "utils.hpp"

namespace my_tensor {
void MyMallocCPU(void** ptr, size_t size) { *ptr = malloc(size); }

void MyMallocGPU(void** ptr, size_t size) { CUDA_CHECK(hipMalloc(ptr, size)); }

void MyMemcpyCPU2CPU(void* dst, const void* src, size_t size) {
  memcpy(dst, src, size);
}

void MyMemcpyCPU2GPU(void* dst, const void* src, size_t size) {
  CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void MyMemcpyGPU2CPU(void* dst, const void* src, size_t size) {
  CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

void MyMemcpyGPU2GPU(void* dst, const void* src, size_t size) {
  CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
}

void MyMemFreeCPU(void* ptr) { free(ptr); }

void MyMemFreeGPU(void* ptr) { CUDA_CHECK(hipFree(ptr)); }
}  // namespace my_tensor
