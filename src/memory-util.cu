#include "hip/hip_runtime.h"
// Copyright 2024 yibotongxue

#include "memory-util.hpp"
#include "utils.hpp"

namespace my_tensor {
void MyMallocCPU(void** ptr, size_t size) { *ptr = malloc(size); }

void MyMallocGPU(void** ptr, size_t size) { CUDA_CHECK(hipMalloc(ptr, size)); }

void MyMemcpyCPU2CPU(void* dst, const void* src, size_t size) {
  memcpy(dst, src, size);
}

void MyMemcpyCPU2GPU(void* dst, const void* src, size_t size) {
  CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

void MyMemcpyGPU2CPU(void* dst, const void* src, size_t size) {
  CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

void MyMemcpyGPU2GPU(void* dst, const void* src, size_t size) {
  CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice));
}

void MyMemFreeCPU(void* ptr) { free(ptr); }

void MyMemFreeGPU(void* ptr) { CUDA_CHECK(hipFree(ptr)); }

namespace {
template <typename T>
__global__ void SetAllValue(T* const data, std::size_t count, T value) {
  CUDA_KERNEL_LOOP(i, count) { data[i] = value; }
}
}  // namespace

template <typename T>
void Fill_GPU(T* const data, std::size_t count, T value) {
  SetAllValue<<<CudaGetBlocks(count), kCudaThreadNum>>>(data, count, value);
}

template void Fill_GPU(float* const data, std::size_t count, float value);
}  // namespace my_tensor
