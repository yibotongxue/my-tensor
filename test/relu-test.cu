#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <relu.cuh>
#include <tensor.cuh>
#include <memory>
#include <vector>
#include <random>

TEST(relu_forward_test, all_positive_on_cpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> input = std::make_shared<my_tensor::Tensor>(shape);
  float *data = input->GetMutableData();
  for (int i = 0; i < 24; i++) {
    *(data + i) = i + 1;
  }
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> output = std::make_shared<my_tensor::Tensor>(shape);
  relu.Forward(input, output);
  const float *output_data = output->GetData();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(output_data + i), i + 1);
  }
}

TEST(relu_forward_test, all_positive_on_gpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> input =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = (float*) malloc(24 * sizeof(float));
  for (int i = 0; i < 24; i++) {
    *(data + i) = i + 1;
  }
  hipMemcpy(input->GetMutableData(), data, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> output =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  relu.Forward(input, output);
  float *output_data = (float*) malloc(24 * sizeof(float));
  hipMemcpy(output_data, output->GetData(), 24 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(output_data + i), i + 1);
  }
  free(output_data);
  output_data = nullptr;
}

TEST(relu_forward_test, all_negative_on_cpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> input = std::make_shared<my_tensor::Tensor>(shape);
  float *data = input->GetMutableData();
  for (int i = 0; i < 24; i++) {
    *(data + i) = -(i + 1);
  }
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> output = std::make_shared<my_tensor::Tensor>(shape);
  relu.Forward(input, output);
  const float *output_data = output->GetData();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(output_data + i), 0);
  }
}

TEST(relu_forward_test, all_negative_on_gpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> input =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = (float*) malloc(24 * sizeof(float));
  for (int i = 0; i < 24; i++) {
    *(data + i) = -(i + 1);
  }
  hipMemcpy(input->GetMutableData(), data, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> output =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  relu.Forward(input, output);
  float *output_data = (float*) malloc(24 * sizeof(float));
  hipMemcpy(output_data, output->GetData(), 24 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(output_data + i), 0);
  }
  free(output_data);
  output_data = nullptr;
}

TEST(relu_forward_test, random_on_cpu) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(-1.0f, 1.0f);
  std::vector<int> shape = { 1000 };
  std::shared_ptr<my_tensor::Tensor> input = std::make_shared<my_tensor::Tensor>(shape);
  float *data = input->GetMutableData();
  for (int i = 0; i < 1000; i++) {
    *(data + i) = dis(gen);
  }
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> output = std::make_shared<my_tensor::Tensor>(shape);
  relu.Forward(input, output);
  const float *output_data = output->GetData();
  for (int i = 0; i < 1000; i++) {
    EXPECT_EQ(*(output_data + i), max(*(data + i), 0.0f));
  }
}

TEST(relu_forward_test, random_on_gpu) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(-1.0f, 1.0f);
  std::vector<int> shape = { 1000 };
  std::shared_ptr<my_tensor::Tensor> input =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = (float*) malloc(1000 * sizeof(float));
  for (int i = 0; i < 1000; i++) {
    *(data + i) = dis(gen);
  }
  hipMemcpy(input->GetMutableData(), data, 1000 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> output =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  relu.Forward(input, output);
  float *output_data = (float*) malloc(1000 * sizeof(float));
  hipMemcpy(output_data, output->GetData(), 1000 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 1000; i++) {
    EXPECT_EQ(*(output_data + i), max(*(data + i), 0.0f));
  }
  free(data);
  free(output_data);
  data = nullptr;
  output_data = nullptr;
}

TEST(relu_backward_test, all_positive_on_cpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom = std::make_shared<my_tensor::Tensor>(shape);
  float *data = bottom->GetMutableData();
  for (int i = 0; i < 24; i++) {
    *(data + i) = i + 1;
  }
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top = std::make_shared<my_tensor::Tensor>(shape);
  relu.Forward(bottom, top);
  float *diff = top->GetMutableDiff();
  for (int i = 0; i < 24; i++) {
    *(diff + i) = i - 12;
  }
  diff = nullptr;
  relu.Backward(top, bottom);
  const float *bottom_diff = bottom->GetDiff();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(bottom_diff + i), i - 12);
  }
  bottom_diff = nullptr;
}

TEST(relu_backward_test, all_positive_on_gpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = (float*) malloc(24 * sizeof(float));
  for (int i = 0; i < 24; i++) {
    *(data + i) = i + 1;
  }
  hipMemcpy(bottom->GetMutableData(), data, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  relu.Forward(bottom, top);
  float *diff = (float*) malloc(24 * sizeof(float));
  for (int i = 0; i < 24; i++) {
    *(diff + i) = i - 12;
  }
  hipMemcpy(top->GetMutableDiff(), diff, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(diff);
  diff = nullptr;
  relu.Backward(top, bottom);
  float *bottom_diff = (float*) malloc(24 * sizeof(float));
  hipMemcpy(bottom_diff, bottom->GetDiff(), 24 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(bottom_diff + i), i - 12);
  }
  free(bottom_diff);
  bottom_diff = nullptr;
}

TEST(relu_backward_test, all_negative_on_cpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom = std::make_shared<my_tensor::Tensor>(shape);
  float *data = bottom->GetMutableData();
  for (int i = 0; i < 24; i++) {
    *(data + i) = -(i + 1);
  }
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top = std::make_shared<my_tensor::Tensor>(shape);
  relu.Forward(bottom, top);
  float *diff = top->GetMutableDiff();
  for (int i = 0; i < 24; i++) {
    *(diff + i) = i - 12;
  }
  diff = nullptr;
  relu.Backward(top, bottom);
  const float *bottom_diff = bottom->GetDiff();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(bottom_diff + i), 0);
  }
  bottom_diff = nullptr;
}

TEST(relu_backward_test, all_negative_on_gpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = (float*) malloc(24 * sizeof(float));
  for (int i = 0; i < 24; i++) {
    *(data + i) = -(i + 1);
  }
  hipMemcpy(bottom->GetMutableData(), data, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  relu.Forward(bottom, top);
  float *diff = (float*) malloc(24 * sizeof(float));
  for (int i = 0; i < 24; i++) {
    *(diff + i) = i - 12;
  }
  hipMemcpy(top->GetMutableDiff(), diff, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(diff);
  diff = nullptr;
  relu.Backward(top, bottom);
  float *bottom_diff = (float*) malloc(24 * sizeof(float));
  hipMemcpy(bottom_diff, bottom->GetDiff(), 24 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(bottom_diff + i), 0);
  }
  free(bottom_diff);
  bottom_diff = nullptr;
}

TEST(relu_backward_test, random_on_cpu) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(-1.0f, 1.0f);
  std::vector<int> shape = { 1000 };
  std::shared_ptr<my_tensor::Tensor> bottom = std::make_shared<my_tensor::Tensor>(shape);
  float *data = bottom->GetMutableData();
  for (int i = 0; i < 1000; i++) {
    *(data + i) = dis(gen);
  }
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top = std::make_shared<my_tensor::Tensor>(shape);
  relu.Forward(bottom, top);
  float *diff = top->GetMutableDiff();
  for (int i = 0; i < 1000; i++) {
    *(diff + i) = dis(gen);
  }
  relu.Backward(top, bottom);
  const float *bottom_diff = bottom->GetDiff();
  for (int i = 0; i < 1000; i++) {
    if (*(data + i) > 0) {
      EXPECT_EQ(*(bottom_diff + i), *(diff + i));
    } else {
      EXPECT_EQ(*(bottom_diff + i), 0);
    }
  }
}

TEST(relu_backward_test, random_on_gpu) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(-1.0f, 1.0f);
  std::vector<int> shape = { 1000 };
  std::shared_ptr<my_tensor::Tensor> bottom =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = (float*) malloc(1000 * sizeof(float));
  for (int i = 0; i < 1000; i++) {
    *(data + i) = dis(gen);
  }
  hipMemcpy(bottom->GetMutableData(), data, 1000 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  relu.Forward(bottom, top);
  float *diff = (float*) malloc(1000 * sizeof(float));
  for (int i = 0; i < 1000; i++) {
    *(diff + i) = dis(gen);
  }
  hipMemcpy(top->GetMutableDiff(), diff, 1000 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  relu.Backward(top, bottom);
  float *bottom_diff = (float*) malloc(1000 * sizeof(float));
  hipMemcpy(bottom_diff, bottom->GetDiff(), 1000 * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < 1000; i++) {
    if (*(data + i) > 0) {
      EXPECT_EQ(*(bottom_diff + i), *(diff + i));
    } else {
      EXPECT_EQ(*(bottom_diff + i), 0);
    }
  }
  free(data);
  free(diff);
  free(bottom_diff);
  data = nullptr;
  diff = nullptr;
  bottom_diff = nullptr;
}

int main(int argc, char** argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
