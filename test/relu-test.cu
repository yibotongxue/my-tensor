#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <relu.cuh>
#include <tensor.cuh>
#include <memory>
#include <vector>
#include <random>

#define TENSOR_CONSTRUCT_ON_CPU(shape_vec, tensor_name) \
  auto tensor_name = std::make_shared<my_tensor::Tensor>(shape_vec);

#define TENSOR_CONSTRUCT_ON_GPU(shape_vec, tensor_name) \
  auto tensor_name = std::make_shared<my_tensor::Tensor>( \
    shape_vec, my_tensor::DeviceType::GPU);

#define DEFINE_DATA_ON_CPU(data_ptr, n, func) \
  float *data_ptr = reinterpret_cast<float*>(malloc(n * sizeof(float))); \
  for (int i = 0; i < n; i++) { \
    data_ptr[i] = func(i); \
  }

#define SET_DATA_ON_CPU(data_ptr, n, func) \
  for (int i = 0; i < n; i++) { \
    data_ptr[i] = func(i); \
  }

#define SET_DATA_ON_GPU(data_ptr, n, func) \
do { \
  DEFINE_DATA_ON_CPU(temp_data, n, func); \
  hipMemcpy(data_ptr, temp_data, n * sizeof(float), hipMemcpyHostToDevice); \
  free(temp_data); \
} while (0);

#define DEFINE_DATA_ON_GPU_FROM_CPU(data_ptr_gpu, data_ptr_cpu, n) \
  float *data_ptr_gpu = nullptr; \
  hipMalloc(&data_ptr_gpu, n * sizeof(float)); \
  hipMemcpy(data_ptr_gpu, data_ptr_cpu, n * sizeof(float), hipMemcpyHostToDevice);

#define DEFINE_DATA_ON_CPU_FROM_GPU(data_ptr_cpu, data_ptr_gpu, n) \
  float *data_ptr_cpu = reinterpret_cast<float*>(malloc(n * sizeof(float))); \
  hipMemcpy(data_ptr_cpu, data_ptr_gpu, n * sizeof(float), hipMemcpyDeviceToHost);

#define SETUP_FOR_RELU_POSITIVE_TEST(device) \
  std::vector<int> shape {1, 2, 3, 4}; \
  TENSOR_CONSTRUCT_ON_##device(shape, bottom); \
  float *data = bottom->GetMutableData(); \
  auto func = [](int x) -> int { return x + 1; }; \
  SET_DATA_ON_##device(data, 24, func); \
  LayerPtr relu = std::make_unique<Relu>(); \
  TENSOR_CONSTRUCT_ON_##device(shape, top); \
  EXPECT_NO_THROW(relu->Forward(bottom, top)); \
  const float *top_data = top->GetData();

TEST(relu_forward_test, all_positive_on_cpu) {
  std::vector<int> shape {1, 2, 3, 4};
  TENSOR_CONSTRUCT_ON_CPU(shape, bottom);
  float *data = bottom->GetMutableData();
  auto func = [](int x) -> int { return x + 1; };
  SET_DATA_ON_CPU(data, 24, func);
  my_tensor::Relu relu;
  TENSOR_CONSTRUCT_ON_CPU(shape, top);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  const float *top_data = top->GetData();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(top_data + i), i + 1);
  }
}

TEST(relu_forward_test, all_positive_on_gpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  for (int i = 0; i < 24; i++) {
    *(data + i) = i + 1;
  }
  hipMemcpy(bottom->GetMutableData(), data, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  float *top_data = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  hipMemcpy(top_data, top->GetData(), 24 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(top_data + i), i + 1);
  }
  free(top_data);
  top_data = nullptr;
}

TEST(relu_forward_test, all_negative_on_cpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom = std::make_shared<my_tensor::Tensor>(shape);
  float *data = bottom->GetMutableData();
  for (int i = 0; i < 24; i++) {
    *(data + i) = -(i + 1);
  }
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top = std::make_shared<my_tensor::Tensor>(shape);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  const float *top_data = top->GetData();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(top_data + i), 0);
  }
}

TEST(relu_forward_test, all_negative_on_gpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  for (int i = 0; i < 24; i++) {
    *(data + i) = -(i + 1);
  }
  hipMemcpy(bottom->GetMutableData(), data, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  float *top_data = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  hipMemcpy(top_data, top->GetData(), 24 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(top_data + i), 0);
  }
  free(top_data);
  top_data = nullptr;
}

TEST(relu_forward_test, random_on_cpu) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(-1.0f, 1.0f);
  std::vector<int> shape = { 10000 };
  std::shared_ptr<my_tensor::Tensor> bottom = std::make_shared<my_tensor::Tensor>(shape);
  float *data = bottom->GetMutableData();
  for (int i = 0; i < 10000; i++) {
    *(data + i) = dis(gen);
  }
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top = std::make_shared<my_tensor::Tensor>(shape);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  const float *top_data = top->GetData();
  for (int i = 0; i < 10000; i++) {
    EXPECT_EQ(*(top_data + i), max(*(data + i), 0.0f));
  }
}

TEST(relu_forward_test, random_on_gpu) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(-1.0f, 1.0f);
  std::vector<int> shape = { 10000 };
  std::shared_ptr<my_tensor::Tensor> bottom =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = reinterpret_cast<float *>(malloc(10000 * sizeof(float)));
  for (int i = 0; i < 10000; i++) {
    *(data + i) = dis(gen);
  }
  hipMemcpy(bottom->GetMutableData(), data, 10000 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  float *top_data = reinterpret_cast<float *>(malloc(10000 * sizeof(float)));
  hipMemcpy(top_data, top->GetData(), 10000 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 10000; i++) {
    EXPECT_EQ(*(top_data + i), max(*(data + i), 0.0f));
  }
  free(data);
  free(top_data);
  data = nullptr;
  top_data = nullptr;
}

TEST(relu_backward_test, all_positive_on_cpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom = std::make_shared<my_tensor::Tensor>(shape);
  float *data = bottom->GetMutableData();
  for (int i = 0; i < 24; i++) {
    *(data + i) = i + 1;
  }
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top = std::make_shared<my_tensor::Tensor>(shape);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  float *diff = top->GetMutableDiff();
  for (int i = 0; i < 24; i++) {
    *(diff + i) = i - 12;
  }
  diff = nullptr;
  EXPECT_NO_THROW(relu.Backward(top, bottom));
  const float *bottom_diff = bottom->GetDiff();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(bottom_diff + i), i - 12);
  }
  bottom_diff = nullptr;
}

TEST(relu_backward_test, all_positive_on_gpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  for (int i = 0; i < 24; i++) {
    *(data + i) = i + 1;
  }
  hipMemcpy(bottom->GetMutableData(), data, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  float *diff = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  for (int i = 0; i < 24; i++) {
    *(diff + i) = i - 12;
  }
  hipMemcpy(top->GetMutableDiff(), diff, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(diff);
  diff = nullptr;
  EXPECT_NO_THROW(relu.Backward(top, bottom));
  float *bottom_diff = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  hipMemcpy(bottom_diff, bottom->GetDiff(), 24 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(bottom_diff + i), i - 12);
  }
  free(bottom_diff);
  bottom_diff = nullptr;
}

TEST(relu_backward_test, all_negative_on_cpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom = std::make_shared<my_tensor::Tensor>(shape);
  float *data = bottom->GetMutableData();
  for (int i = 0; i < 24; i++) {
    *(data + i) = -(i + 1);
  }
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top = std::make_shared<my_tensor::Tensor>(shape);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  float *diff = top->GetMutableDiff();
  for (int i = 0; i < 24; i++) {
    *(diff + i) = i - 12;
  }
  diff = nullptr;
  EXPECT_NO_THROW(relu.Backward(top, bottom));
  const float *bottom_diff = bottom->GetDiff();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(bottom_diff + i), 0);
  }
  bottom_diff = nullptr;
}

TEST(relu_backward_test, all_negative_on_gpu) {
  std::vector<int> shape {1, 2, 3, 4};
  std::shared_ptr<my_tensor::Tensor> bottom =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  for (int i = 0; i < 24; i++) {
    *(data + i) = -(i + 1);
  }
  hipMemcpy(bottom->GetMutableData(), data, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  float *diff = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  for (int i = 0; i < 24; i++) {
    *(diff + i) = i - 12;
  }
  hipMemcpy(top->GetMutableDiff(), diff, 24 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(diff);
  diff = nullptr;
  EXPECT_NO_THROW(relu.Backward(top, bottom));
  float *bottom_diff = reinterpret_cast<float *>(malloc(24 * sizeof(float)));
  hipMemcpy(bottom_diff, bottom->GetDiff(), 24 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 24; i++) {
    EXPECT_EQ(*(bottom_diff + i), 0);
  }
  free(bottom_diff);
  bottom_diff = nullptr;
}

TEST(relu_backward_test, random_on_cpu) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(-1.0f, 1.0f);
  std::vector<int> shape = { 10000 };
  std::shared_ptr<my_tensor::Tensor> bottom = std::make_shared<my_tensor::Tensor>(shape);
  float *data = bottom->GetMutableData();
  for (int i = 0; i < 10000; i++) {
    *(data + i) = dis(gen);
  }
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top = std::make_shared<my_tensor::Tensor>(shape);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  float *diff = top->GetMutableDiff();
  for (int i = 0; i < 10000; i++) {
    *(diff + i) = dis(gen);
  }
  EXPECT_NO_THROW(relu.Backward(top, bottom));
  const float *bottom_diff = bottom->GetDiff();
  for (int i = 0; i < 10000; i++) {
    if (*(data + i) > 0) {
      EXPECT_EQ(*(bottom_diff + i), *(diff + i));
    } else {
      EXPECT_EQ(*(bottom_diff + i), 0);
    }
  }
}

TEST(relu_backward_test, random_on_gpu) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(-1.0f, 1.0f);
  std::vector<int> shape = { 10000 };
  std::shared_ptr<my_tensor::Tensor> bottom =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  float *data = reinterpret_cast<float *>(malloc(10000 * sizeof(float)));
  for (int i = 0; i < 10000; i++) {
    *(data + i) = dis(gen);
  }
  hipMemcpy(bottom->GetMutableData(), data, 10000 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  my_tensor::Relu relu;
  std::shared_ptr<my_tensor::Tensor> top =
    std::make_shared<my_tensor::Tensor>(shape, my_tensor::DeviceType::GPU);
  EXPECT_NO_THROW(relu.Forward(bottom, top));
  float *diff = reinterpret_cast<float *>(malloc(10000 * sizeof(float)));
  for (int i = 0; i < 10000; i++) {
    *(diff + i) = dis(gen);
  }
  hipMemcpy(top->GetMutableDiff(), diff, 10000 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  EXPECT_NO_THROW(relu.Backward(top, bottom));
  float *bottom_diff = reinterpret_cast<float *>(malloc(10000 * sizeof(float)));
  hipMemcpy(bottom_diff, bottom->GetDiff(), 10000 * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < 10000; i++) {
    if (*(data + i) > 0) {
      EXPECT_EQ(*(bottom_diff + i), *(diff + i));
    } else {
      EXPECT_EQ(*(bottom_diff + i), 0);
    }
  }
  free(data);
  free(diff);
  free(bottom_diff);
  data = nullptr;
  diff = nullptr;
  bottom_diff = nullptr;
}

int main(int argc, char** argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
