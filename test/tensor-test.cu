#include <gtest/gtest.h>
#include <tensor.cuh>


/*************************TENSOR_TEST_CONSTRUCT**************************** */
TEST(tensor_test_construct, tensor_test_construct_shape_cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape };
  EXPECT_EQ(tensor.GetShape(), shape);
}

TEST(tensor_test_construct, tensor_test_construct_shape_gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  EXPECT_EQ(tensor.GetShape(), shape);
}

TEST(tensor_test_construct, tensor_test_construct_device_cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape };
  EXPECT_TRUE(tensor.OnCPU());
  EXPECT_FALSE(tensor.OnGPU());
}

TEST(tensor_test_construct, tensor_test_construct_device_explicit_cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::CPU };
  EXPECT_TRUE(tensor.OnCPU());
  EXPECT_FALSE(tensor.OnGPU());
}

TEST(tensor_test_construct, tensor_test_construct_device_gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  EXPECT_FALSE(tensor.OnCPU());
  EXPECT_TRUE(tensor.OnGPU());
}

TEST(tensor_test_construct, tensor_test_construct_data_position_cpu) {
  std::vector<int> shape { 1, 3, 2 };
  my_tensor::Tensor tensor { shape };
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(data, tensor.GetData(), 6 * sizeof(float), hipMemcpyHostToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_construct, tensor_test_construct_data_position_gpu) {
  std::vector<int> shape { 1, 3, 2 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error = 
    hipMemcpy(data, tensor.GetData(), 6 * sizeof(float), hipMemcpyDeviceToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}
/*************************TENSOR_TEST_CONSTRUCT**************************** */



/**********************TENSOR_TEST_COPY_CONSTRUCT************************** */
TEST(tensor_test_copy_construct, tensor_test_copy_construct_shape_cpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { tensor };
  EXPECT_EQ(another.GetShape(), shape);
}

TEST(tensor_test_copy_construct, tensor_test_copy_construct_shape_gpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { tensor };
  EXPECT_EQ(another.GetShape(), shape);
}

TEST(tensor_test_copy_construct, tensor_test_copy_construct_device_cpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { tensor };
  EXPECT_TRUE(another.OnCPU());
  EXPECT_FALSE(another.OnGPU());
}

TEST(tensor_test_copy_construct, tensor_test_copy_construct_device_explicit_cpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::CPU };
  my_tensor::Tensor another { tensor };
  EXPECT_TRUE(another.OnCPU());
  EXPECT_FALSE(another.OnGPU());
}

TEST(tensor_test_copy_construct, tensor_test_copy_construct_device_gpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { tensor };
  EXPECT_FALSE(another.OnCPU());
  EXPECT_TRUE(another.OnGPU());
}

TEST(tensor_test_copy_construct, tensor_test_copy_construct_data_position_cpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { tensor };
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyHostToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_copy_construct, tensor_test_copy_construct_data_position_gpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { tensor };
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error = 
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_copy_construct, tensor_test_copy_construct_data_cpu2cpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape };
  float* data = tensor.GetMutableData();
  for (int i = 0; i < 6; ++i) {
    *(data + i) = static_cast<float>(i);
  }
  my_tensor::Tensor another { tensor };
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another.GetData() + i), static_cast<float>(i));
  }
}

TEST(tensor_test_copy_construct, tensor_test_copy_construct_data_gpu2gpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  float* data = (float*) malloc(6 * sizeof(float));
  for (int i = 0; i < 6; ++i) {
    *(data + i) = (float)i;
  }
  hipMemcpy(tensor.GetMutableData(), data, 6 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Tensor another { tensor };
  float *another_data = (float*) malloc(6 * sizeof(float));
  hipMemcpy(another_data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another_data + i), static_cast<float>(i));
  }
  free(another_data);
}
/**********************TENSOR_TEST_COPY_CONSTRUCT************************** */



/****************************TENSOR_TEST_COPY****************************** */
TEST(tensor_test_copy, tensor_test_copy_shape) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { another_shape };
  another = tensor;
  EXPECT_EQ(another.GetShape(), shape);
}

TEST(tensor_test_copy, tensor_test_copy_device_cpu2cpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { another_shape };
  another = tensor;
  EXPECT_TRUE(another.OnCPU());
  EXPECT_FALSE(another.OnGPU());
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyHostToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_copy, tensor_test_copy_device_cpu2gpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = tensor;
  EXPECT_TRUE(another.OnCPU());
  EXPECT_FALSE(another.OnGPU());
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyHostToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_copy, tensor_test_copy_device_gpu2cpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { another_shape };
  another = tensor;
  EXPECT_TRUE(another.OnGPU());
  EXPECT_FALSE(another.OnCPU());
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error = 
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_copy, tensor_test_copy_device_gpu2gpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = tensor;
  EXPECT_TRUE(another.OnGPU());
  EXPECT_FALSE(another.OnCPU());
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error = 
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_copy, tensor_test_copy_data_cpu2cpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  float* data = tensor.GetMutableData();
  for (int i = 0; i < 6; ++i) {
    *(data + i) = static_cast<float>(i);
  }
  my_tensor::Tensor another { another_shape };
  another = tensor;
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another.GetData() + i), static_cast<float>(i));
  }
}

TEST(tensor_test_copy, tensor_test_copy_data_cpu2gpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  float* data = tensor.GetMutableData();
  for (int i = 0; i < 6; ++i) {
    *(data + i) = static_cast<float>(i);
  }
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = tensor;
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another.GetData() + i), static_cast<float>(i));
  }
}

TEST(tensor_test_copy, tensor_test_copy_data_gpu2cpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  float* data = (float*) malloc(6 * sizeof(float));
  for (int i = 0; i < 6; ++i) {
    *(data + i) = (float)i;
  }
  hipMemcpy(tensor.GetMutableData(), data, 6 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Tensor another { another_shape };
  another = tensor;
  float *another_data = (float*) malloc(6 * sizeof(float));
  hipMemcpy(another_data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another_data + i), static_cast<float>(i));
  }
  free(another_data);
}

TEST(tensor_test_copy, tensor_test_copy_data_gpu2gpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  float* data = (float*) malloc(6 * sizeof(float));
  for (int i = 0; i < 6; ++i) {
    *(data + i) = (float)i;
  }
  hipMemcpy(tensor.GetMutableData(), data, 6 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = tensor;
  float *another_data = (float*) malloc(6 * sizeof(float));
  hipMemcpy(another_data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another_data + i), static_cast<float>(i));
  }
  free(another_data);
}
/****************************TENSOR_TEST_COPY****************************** */


/***********************TENSOR_TEST_MOVE_CONSTRUCT************************* */
TEST(tensor_test_move_construct, tensor_test_move_construct_cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape };
  const float *data = tensor.GetData();
  my_tensor::Tensor another { std::move(tensor) };
  EXPECT_EQ(another.GetData(), data);
  EXPECT_EQ(tensor.GetData(), nullptr);
}

TEST(tensor_test_move_construct, tensor_test_move_construct_gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  const float *data = tensor.GetData();
  my_tensor::Tensor another { std::move(tensor) };
  EXPECT_EQ(another.GetData(), data);
  EXPECT_EQ(tensor.GetData(), nullptr);
}

TEST(tensor_test_move_construct, tensor_test_move_construct_shape_cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { std::move(tensor) };
  EXPECT_EQ(another.GetShape(), shape);
}

TEST(tensor_test_move_construct, tensor_test_move_construct_shape_gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { std::move(tensor) };
  EXPECT_EQ(another.GetShape(), shape);
}

TEST(tensor_test_move_construct, tensor_test_move_construct_device_cpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { std::move(tensor) };
  EXPECT_TRUE(another.OnCPU());
  EXPECT_FALSE(another.OnGPU());
}

TEST(tensor_test_move_construct, tensor_test_move_construct_device_explicit_cpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::CPU };
  my_tensor::Tensor another { std::move(tensor) };
  EXPECT_TRUE(another.OnCPU());
  EXPECT_FALSE(another.OnGPU());
}

TEST(tensor_test_move_construct, tensor_test_move_construct_device_gpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { std::move(tensor) };
  EXPECT_FALSE(another.OnCPU());
  EXPECT_TRUE(another.OnGPU());
}

TEST(tensor_test_move_construct, tensor_test_move_construct_data_position_cpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { std::move(tensor) };
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyHostToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_move_construct, tensor_test_move_construct_data_position_gpu) {
  std::vector<int> shape { 1, 2, 3 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { std::move(tensor) };
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error = 
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}
/***********************TENSOR_TEST_MOVE_CONSTRUCT************************* */



/***************************TENSOR_TEST_MOVE******************************* */
TEST(tensor_test_move, tensor_test_move_cpu2cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape };
  const float *data = tensor.GetData();
  std::vector<int> another_shape {2, 3, 4};
  my_tensor::Tensor another { another_shape };
  another = std::move(tensor);
  EXPECT_EQ(another.GetData(), data);
  EXPECT_EQ(tensor.GetData(), nullptr);
}

TEST(tensor_test_move, tensor_test_move_cpu2gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape };
  const float *data = tensor.GetData();
  std::vector<int> another_shape {2, 3, 4};
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = std::move(tensor);
  EXPECT_EQ(another.GetData(), data);
  EXPECT_EQ(tensor.GetData(), nullptr);
}

TEST(tensor_test_move, tensor_test_move_gpu2cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  const float *data = tensor.GetData();
  std::vector<int> another_shape {2, 3, 4};
  my_tensor::Tensor another { another_shape };
  another = std::move(tensor);
  EXPECT_EQ(another.GetData(), data);
  EXPECT_EQ(tensor.GetData(), nullptr);
}

TEST(tensor_test_move, tensor_test_move_gpu2gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  const float *data = tensor.GetData();
  std::vector<int> another_shape {2, 3, 4};
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = std::move(tensor);
  EXPECT_EQ(another.GetData(), data);
  EXPECT_EQ(tensor.GetData(), nullptr);
}

TEST(tensor_test_move, tensor_test_move_shape) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { another_shape };
  another = std::move(tensor);
  EXPECT_EQ(another.GetShape(), shape);
}

TEST(tensor_test_move, tensor_test_move_device_cpu2cpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { another_shape };
  another = std::move(tensor);
  EXPECT_TRUE(another.OnCPU());
  EXPECT_FALSE(another.OnGPU());
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyHostToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_move, tensor_test_move_device_cpu2gpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = std::move(tensor);
  EXPECT_TRUE(another.OnCPU());
  EXPECT_FALSE(another.OnGPU());
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyHostToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_move, tensor_test_move_device_gpu2cpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { another_shape };
  another = std::move(tensor);
  EXPECT_TRUE(another.OnGPU());
  EXPECT_FALSE(another.OnCPU());
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error = 
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_move, tensor_test_move_device_gpu2gpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = std::move(tensor);
  EXPECT_TRUE(another.OnGPU());
  EXPECT_FALSE(another.OnCPU());
  float* data = nullptr;
  hipMalloc(&data, 6 * sizeof(float));
  hipError_t error = 
    hipMemcpy(data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(data);
  hipDeviceSynchronize();
}

TEST(tensor_test_move, tensor_test_move_data_cpu2cpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  float* data = tensor.GetMutableData();
  for (int i = 0; i < 6; ++i) {
    *(data + i) = static_cast<float>(i);
  }
  my_tensor::Tensor another { another_shape };
  another = std::move(tensor);
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another.GetData() + i), static_cast<float>(i));
  }
}

TEST(tensor_test_move, tensor_test_move_data_cpu2gpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape };
  float* data = tensor.GetMutableData();
  for (int i = 0; i < 6; ++i) {
    *(data + i) = static_cast<float>(i);
  }
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = std::move(tensor);
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another.GetData() + i), static_cast<float>(i));
  }
}

TEST(tensor_test_move, tensor_test_move_data_gpu2cpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  float* data = (float*) malloc(6 * sizeof(float));
  for (int i = 0; i < 6; ++i) {
    *(data + i) = (float)i;
  }
  hipMemcpy(tensor.GetMutableData(), data, 6 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Tensor another { another_shape };
  another = std::move(tensor);
  float *another_data = (float*) malloc(6 * sizeof(float));
  hipMemcpy(another_data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another_data + i), static_cast<float>(i));
  }
  free(another_data);
}

TEST(tensor_test_move, tensor_test_move_data_gpu2gpu) {
  std::vector<int> shape { 1, 2, 3 };
  std::vector<int> another_shape { 3, 2, 4 };
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  float* data = (float*) malloc(6 * sizeof(float));
  for (int i = 0; i < 6; ++i) {
    *(data + i) = (float)i;
  }
  hipMemcpy(tensor.GetMutableData(), data, 6 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  data = nullptr;
  my_tensor::Tensor another { another_shape, my_tensor::DeviceType::GPU };
  another = std::move(tensor);
  float *another_data = (float*) malloc(6 * sizeof(float));
  hipMemcpy(another_data, another.GetData(), 6 * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(another_data + i), static_cast<float>(i));
  }
  free(another_data);
}
/***************************TENSOR_TEST_MOVE******************************* */



/****************************TENSOR_TEST_CPU******************************* */
TEST(tensor_test_cpu, tensor_test_cpu_on_cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::CPU };
  my_tensor::Tensor cpu_tensor = tensor.cpu();
  EXPECT_EQ(cpu_tensor.GetShape(), shape);
  EXPECT_TRUE(cpu_tensor.OnCPU());
  EXPECT_FALSE(cpu_tensor.OnGPU());
}

TEST(tensor_test_cpu, tensor_test_cpu_on_gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor cpu_tensor = tensor.cpu();
  EXPECT_EQ(cpu_tensor.GetShape(), shape);
  EXPECT_TRUE(cpu_tensor.OnCPU());
  EXPECT_FALSE(cpu_tensor.OnGPU());
}

TEST(tensor_test_cpu, tensor_test_cpu_data_on_cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::CPU };
  float *data = tensor.GetMutableData();
  for (int i = 0; i < 6; ++i) {
    *(data + i) = i + 1;
  }
  my_tensor::Tensor cpu_tensor = tensor.cpu();
  float *gpu_data = nullptr;
  hipMalloc(&gpu_data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(gpu_data, cpu_tensor.GetData(), 6 * sizeof(float), hipMemcpyHostToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(gpu_data);
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(cpu_tensor.GetData() + i), i + 1);
  }
  hipDeviceSynchronize();
}

TEST(tensor_test_cpu, tensor_test_cpu_data_on_gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  float *data = (float*) malloc(6 * sizeof(float));
  for (int i = 0; i < 6; ++i) {
    *(data + i) = i + 1;
  }
  hipMemcpy(tensor.GetMutableData(), data, 6 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  my_tensor::Tensor cpu_tensor = tensor.cpu();
  float *gpu_data = nullptr;
  hipMalloc(&gpu_data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(gpu_data, cpu_tensor.GetData(), 6 * sizeof(float), hipMemcpyHostToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(gpu_data);
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(cpu_tensor.GetData() + i), i + 1);
  }
  hipDeviceSynchronize();
}
/****************************TENSOR_TEST_CPU******************************* */


/****************************TENSOR_TEST_GPU******************************* */
TEST(tensor_test_gpu, tensor_test_gpu_on_cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::CPU };
  my_tensor::Tensor gpu_tensor = tensor.gpu();
  EXPECT_EQ(gpu_tensor.GetShape(), shape);
  EXPECT_TRUE(gpu_tensor.OnGPU());
  EXPECT_FALSE(gpu_tensor.OnCPU());
}

TEST(tensor_test_gpu, tensor_test_gpu_on_gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  my_tensor::Tensor gpu_tensor = tensor.gpu();
  EXPECT_EQ(gpu_tensor.GetShape(), shape);
  EXPECT_TRUE(gpu_tensor.OnGPU());
  EXPECT_FALSE(gpu_tensor.OnCPU());
}

TEST(tensor_test_gpu, tensor_test_gpu_data_on_cpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::CPU };
  float *data = tensor.GetMutableData();
  for (int i = 0; i < 6; ++i) {
    *(data + i) = i + 1;
  }
  my_tensor::Tensor gpu_tensor = tensor.gpu();
  float *gpu_data = nullptr;
  hipMalloc(&gpu_data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(gpu_data, gpu_tensor.GetData(), 6 * sizeof(float), hipMemcpyDeviceToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(gpu_data);
  float *cpu_data = (float*) malloc(6 * sizeof(float));
  hipMemcpy(cpu_data, gpu_tensor.GetData(), 6 * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(cpu_data + i), i + 1);
  }
  free(cpu_data);
  hipDeviceSynchronize();
}

TEST(tensor_test_gpu, tensor_test_gpu_data_on_gpu) {
  std::vector<int> shape {1, 2, 3};
  my_tensor::Tensor tensor { shape, my_tensor::DeviceType::GPU };
  float *data = (float*) malloc(6 * sizeof(float));
  for (int i = 0; i < 6; ++i) {
    *(data + i) = i + 1;
  }
  hipMemcpy(tensor.GetMutableData(), data, 6 * sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  free(data);
  my_tensor::Tensor gpu_tensor = tensor.gpu();
  float *gpu_data = nullptr;
  hipMalloc(&gpu_data, 6 * sizeof(float));
  hipError_t error =
    hipMemcpy(gpu_data, gpu_tensor.GetData(), 6 * sizeof(float), hipMemcpyDeviceToDevice);
  EXPECT_EQ(error, hipSuccess);
  hipFree(gpu_data);
  float *cpu_data = (float*) malloc(6 * sizeof(float));
  hipMemcpy(cpu_data, gpu_tensor.GetData(), 6 * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < 6; ++i) {
    EXPECT_EQ(*(cpu_data + i), i + 1);
  }
  free(cpu_data);
  hipDeviceSynchronize();
}
/****************************TENSOR_TEST_GPU******************************* */



int main(int argc, char** argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
