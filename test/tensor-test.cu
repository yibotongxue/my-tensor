#include <gtest/gtest.h>
#include <tensor.cuh>
#include <functional>

#define TENSOR_CONSTRUCT_ON_CPU(shape_vec, tensor_name) \
  auto tensor_name = std::make_unique<my_tensor::Tensor>(shape_vec);

#define TENSOR_CONSTRUCT_ON_GPU(shape_vec, tensor_name) \
  auto tensor_name = std::make_unique<my_tensor::Tensor>( \
    shape_vec, my_tensor::DeviceType::GPU);

#define TENSOR_CONSTRUCTOR_COPY(tensor_dst, tensor_src) \
  std::unique_ptr<my_tensor::Tensor> tensor_dst = std::make_unique<my_tensor::Tensor>(*tensor_src);

#define TENSOR_CONSTRUCTOR_MOVE(tensor_dst, tensor_src) \
  std::unique_ptr<my_tensor::Tensor> tensor_dst = std::make_unique<my_tensor::Tensor>(std::move(*tensor_src));

#define TENSOR_EXPECT_SHAPE(tensor_ptr, shape_vec) \
do { \
  EXPECT_EQ(tensor_ptr->GetShape(), shape_vec); \
} while (0);

#define TENSOR_EXPECT_ON_CPU(tensor_ptr) \
do { \
  EXPECT_TRUE(tensor_ptr->OnCPU()); \
  EXPECT_FALSE(tensor_ptr->OnGPU()); \
} while (0);

#define TENSOR_EXPECT_ON_GPU(tensor_ptr) \
do { \
  EXPECT_TRUE(tensor_ptr->OnGPU()); \
  EXPECT_FALSE(tensor_ptr->OnCPU()); \
} while (0);

#define TENSOR_DATA_ON_CPU(tensor_ptr) \
do { \
  std::size_t byte_size = tensor_ptr->GetByteSize(); \
  float* data = nullptr; \
  hipMalloc(&data, byte_size); \
  hipError_t error = \
    hipMemcpy(data, tensor_ptr->GetData(), byte_size, hipMemcpyHostToDevice); \
  EXPECT_EQ(error, hipSuccess); \
  hipFree(data); \
} while (0);

#define TENSOR_DATA_ON_GPU(tensor_ptr) \
do { \
  std::size_t byte_size = tensor_ptr->GetByteSize(); \
  float* data = nullptr; \
  hipMalloc(&data, byte_size); \
  hipError_t error = \
    hipMemcpy(data, tensor_ptr->GetData(), byte_size, hipMemcpyDeviceToDevice); \
  EXPECT_EQ(error, hipSuccess); \
  hipFree(data); \
} while (0);

#define DATA_EXPECT_EQ(data1, data2, n) \
do { \
  for (int i = 0; i < n; i++) { \
    EXPECT_EQ(data1[i], data2[i]); \
  } \
} while (0);

#define DEFINE_DATA_ON_CPU(data_ptr, n, func) \
  float *data_ptr = reinterpret_cast<float*>(malloc(n * sizeof(float))); \
  for (int i = 0; i < n; i++) { \
    data_ptr[i] = func(i); \
  }

#define SET_DATA_ON_CPU(data_ptr, n, func) \
  for (int i = 0; i < n; i++) { \
    data_ptr[i] = func(i); \
  }

#define SET_DATA_ON_GPU(data_ptr, n, func) \
do { \
  DEFINE_DATA_ON_CPU(temp_data, n, func); \
  hipMemcpy(data_ptr, temp_data, n * sizeof(float), hipMemcpyHostToDevice); \
  free(temp_data); \
} while (0);

#define DEFINE_DATA_ON_GPU_FROM_CPU(data_ptr_gpu, data_ptr_cpu, n) \
  float *data_ptr_gpu = nullptr; \
  hipMalloc(&data_ptr_gpu, n * sizeof(float)); \
  hipMemcpy(data_ptr_gpu, data_ptr_cpu, n * sizeof(float), hipMemcpyHostToDevice);

#define DEFINE_DATA_ON_CPU_FROM_GPU(data_ptr_cpu, data_ptr_gpu, n) \
  float *data_ptr_cpu = reinterpret_cast<float*>(malloc(n * sizeof(float))); \
  hipMemcpy(data_ptr_cpu, data_ptr_gpu, n * sizeof(float), hipMemcpyDeviceToHost);

#define TENSOR_EXPECT_EQ_DATA_CPU_CPU(tensor_this, tensor_that) \
do { \
  int n = tensor_this->GetSize(); \
  EXPECT_EQ(tensor_that->GetSize(), n); \
  DATA_EXPECT_EQ(tensor_this->GetData(), tensor_that->GetData(), n); \
} while (0);

#define TENSOR_EXPECT_EQ_DATA_CPU_GPU(tensor_this, tensor_that) \
do { \
  int n = tensor_this->GetSize(); \
  EXPECT_EQ(tensor_that->GetSize(), n); \
  DEFINE_DATA_ON_CPU_FROM_GPU(data_that, tensor_that->GetData(), n); \
  DATA_EXPECT_EQ(tensor_this->GetData(), data_that, n); \
  free(data_that); \
} while (0);

#define TENSOR_EXPECT_EQ_DATA_GPU_CPU(tensor_this, tensor_that) \
do { \
  int n = tensor_this->GetSize(); \
  EXPECT_EQ(tensor_that->GetSize(), n); \
  DEFINE_DATA_ON_CPU_FROM_GPU(data_this, tensor_this->GetData(), n); \
  DATA_EXPECT_EQ(data_this, tensor_that->GetData(), n); \
  free(data_this); \
} while (0);

#define TENSOR_EXPECT_EQ_DATA_GPU_GPU(tensor_this, tensor_that) \
do { \
  int n = tensor_this->GetSize(); \
  EXPECT_EQ(tensor_that->GetSize(), n); \
  DEFINE_DATA_ON_CPU_FROM_GPU(data_this, tensor_this->GetData(), n); \
  DEFINE_DATA_ON_CPU_FROM_GPU(data_that, tensor_that->GetData(), n); \
  DATA_EXPECT_EQ(data_this, data_that, n); \
  free(data_this); \
  free(data_that); \
} while (0);

// define basic test of shape, oncpu and ongpu, and data position
#define TENSOR_SHAPE_TEST(common, device) \
TEST_F(common##device, shape_test) { \
  TENSOR_EXPECT_SHAPE(tensor, shape); \
}

#define TENSOR_ON_DEVICE_TEST(common, device) \
TEST_F(common##device, ondevice_test) { \
  TENSOR_EXPECT_ON_##device(tensor); \
}

#define TENSOR_DATA_ON_DEVICE_TEST(common, device) \
TEST_F(common##device, data_ondevice_##device) { \
  TENSOR_DATA_ON_##device(tensor); \
}

#define TENSOR_CONSTRUCT_BASIC_TEST_SPECIFIC_DEVICE(common, device) \
  TENSOR_SHAPE_TEST(common, device) \
  TENSOR_ON_DEVICE_TEST(common, device) \
  TENSOR_DATA_ON_DEVICE_TEST(common, device)

#define TENSOR_CONSTRUCT_BASIC_TEST(common) \
  TENSOR_CONSTRUCT_BASIC_TEST_SPECIFIC_DEVICE(common, CPU) \
  TENSOR_CONSTRUCT_BASIC_TEST_SPECIFIC_DEVICE(common, GPU)


// define basic test of shape, oncpu and ongpu, and data position of two devices
#define TENSOR_SHAPE_TWO_DEVICES_TEST(common, device_from, device_to) \
TEST_F(common##device_from##2##device_to, shape_test) { \
  TENSOR_EXPECT_SHAPE(tensor, shape); \
}

#define TENSOR_ON_DEVICE_TWO_DEVICES_TEST(common, device_from, device_to) \
TEST_F(common##device_from##2##device_to, ondevice_test) { \
  TENSOR_EXPECT_ON_##device_from(tensor); \
}

#define TENSOR_DATA_ON_DEVICE_TWO_DEVICES_TEST(common, device_from, device_to) \
TEST_F(common##device_from##2##device_to, data_ondevice) { \
  TENSOR_DATA_ON_##device_from(tensor); \
}

#define TENSOR_MOVE_OR_COPY_BASIC_TEST_SPECIFIC_DEVICES(common, device_from, device_to) \
  TENSOR_SHAPE_TWO_DEVICES_TEST(common, device_from, device_to) \
  TENSOR_ON_DEVICE_TWO_DEVICES_TEST(common, device_from, device_to) \
  TENSOR_DATA_ON_DEVICE_TWO_DEVICES_TEST(common, device_from, device_to)

#define TENSOR_MOVE_OR_COPY_BASIC_TEST(common) \
  TENSOR_MOVE_OR_COPY_BASIC_TEST_SPECIFIC_DEVICES(common, CPU, CPU) \
  TENSOR_MOVE_OR_COPY_BASIC_TEST_SPECIFIC_DEVICES(common, CPU, GPU) \
  TENSOR_MOVE_OR_COPY_BASIC_TEST_SPECIFIC_DEVICES(common, GPU, CPU) \
  TENSOR_MOVE_OR_COPY_BASIC_TEST_SPECIFIC_DEVICES(common, GPU, GPU)

/*************************TENSOR_TEST_CONSTRUCT**************************** */
#define TENSOR_CONSTRUCT_TEST_CLASS(device) \
  class TensorConstructTest##device : public ::testing::Test { \
   protected: \
    void SetUp() override { \
      tensor = \
        std::move(std::make_unique<my_tensor::Tensor>(shape, my_tensor::DeviceType::device)); \
    } \
    std::vector<int> shape {1, 2, 3}; \
    std::unique_ptr<my_tensor::Tensor> tensor; \
  };

TENSOR_CONSTRUCT_TEST_CLASS(CPU);
TENSOR_CONSTRUCT_TEST_CLASS(GPU);

TENSOR_CONSTRUCT_BASIC_TEST(TensorConstructTest)
/*************************TENSOR_TEST_CONSTRUCT**************************** */



/**********************TENSOR_TEST_COPY_CONSTRUCT************************** */
#define TENSOR_COPY_CONSTRUCT_TEST_CLASS(device) \
  class TensorCopyConstructTest##device : public ::testing::Test { \
   protected: \
    void SetUp() override { \
      another = \
        std::move(std::make_unique<my_tensor::Tensor>(shape, my_tensor::DeviceType::device)); \
      auto func = [](int x) -> float { return x; }; \
      float *data = another->GetMutableData(); \
      SET_DATA_ON_##device(data, 6, func); \
      tensor = \
        std::move(std::make_unique<my_tensor::Tensor>(*another)); \
    } \
    std::vector<int> shape {1, 2, 3}; \
    std::unique_ptr<my_tensor::Tensor> another; \
    std::unique_ptr<my_tensor::Tensor> tensor; \
  };

TENSOR_COPY_CONSTRUCT_TEST_CLASS(CPU);
TENSOR_COPY_CONSTRUCT_TEST_CLASS(GPU);

TENSOR_CONSTRUCT_BASIC_TEST(TensorCopyConstructTest)

#define TENSOR_COPY_CONSTRUCT_SUCCESSFULLY(device) \
TEST_F(TensorCopyConstructTest##device, data_copy) { \
  TENSOR_EXPECT_EQ_DATA_##device##_##device(another, tensor); \
}

TENSOR_COPY_CONSTRUCT_SUCCESSFULLY(CPU)
TENSOR_COPY_CONSTRUCT_SUCCESSFULLY(GPU)
/**********************TENSOR_TEST_COPY_CONSTRUCT************************** */


/***********************TENSOR_TEST_MOVE_CONSTRUCT************************* */
#define TENSOR_MOVE_CONSTRUCT_TEST_CLASS(device) \
  class TensorMoveConstructTest##device : public ::testing::Test { \
   protected: \
    void SetUp() override { \
      another = \
        std::move(std::make_unique<my_tensor::Tensor>(shape, my_tensor::DeviceType::device)); \
      auto func = [](int x) -> float { return x; }; \
      float *data = another->GetMutableData(); \
      SET_DATA_ON_##device(data, 6, func); \
      temp_data = another->GetData(); \
      tensor = \
        std::move(std::make_unique<my_tensor::Tensor>(std::move(*another))); \
    } \
    std::vector<int> shape {1, 2, 3}; \
    std::unique_ptr<my_tensor::Tensor> another; \
    std::unique_ptr<my_tensor::Tensor> tensor; \
    const float *temp_data; \
  };

TENSOR_MOVE_CONSTRUCT_TEST_CLASS(CPU)
TENSOR_MOVE_CONSTRUCT_TEST_CLASS(GPU)

TENSOR_CONSTRUCT_BASIC_TEST(TensorMoveConstructTest)

#define TENSOR_MOVE_CONSTRUCT_SUCCESSFULLY(device) \
TEST_F(TensorMoveConstructTest##device, data_move) { \
  EXPECT_EQ(tensor->GetData(), temp_data); \
  EXPECT_EQ(another->GetData(), nullptr); \
}

TENSOR_MOVE_CONSTRUCT_SUCCESSFULLY(CPU)
TENSOR_MOVE_CONSTRUCT_SUCCESSFULLY(GPU)

#define TENSOR_MOVE_SUCCESSFULY \
do { \
  EXPECT_EQ(another_data, temp_data); \
  EXPECT_EQ(tensor_data, nullptr); \
} while (0);
/***********************TENSOR_TEST_MOVE_CONSTRUCT************************* */



/****************************TENSOR_TEST_COPY****************************** */
#define TENSOR_COPY_TEST_CLASS(device_from, device_to) \
  class TensorCopyTest##device_from##2##device_to : public ::testing::Test { \
   protected: \
    void SetUp() override { \
      another = \
        std::move(std::make_unique<my_tensor::Tensor>(shape, my_tensor::DeviceType::device_from)); \
      auto func = [](int x) -> float { return x; }; \
      float *data = another->GetMutableData(); \
      SET_DATA_ON_##device_from(data, 6, func); \
      tensor = \
        std::move(std::make_unique<my_tensor::Tensor>(other_shape, my_tensor::DeviceType::device_to)); \
      *tensor = *another; \
    } \
    std::vector<int> shape {1, 2, 3}; \
    std::vector<int> other_shape {2, 3, 4}; \
    std::unique_ptr<my_tensor::Tensor> another; \
    std::unique_ptr<my_tensor::Tensor> tensor; \
  };

TENSOR_COPY_TEST_CLASS(CPU, CPU);
TENSOR_COPY_TEST_CLASS(CPU, GPU);
TENSOR_COPY_TEST_CLASS(GPU, CPU);
TENSOR_COPY_TEST_CLASS(GPU, GPU);

TENSOR_MOVE_OR_COPY_BASIC_TEST(TensorCopyTest)

#define TENSOR_COPY_SUCCESSFULLY(device_from, device_to) \
TEST_F(TensorCopyTest##device_from##2##device_to, data_copy) { \
  TENSOR_EXPECT_EQ_DATA_##device_from##_##device_from(another, tensor); \
}

TENSOR_COPY_SUCCESSFULLY(CPU, CPU);
TENSOR_COPY_SUCCESSFULLY(CPU, GPU);
TENSOR_COPY_SUCCESSFULLY(GPU, CPU);
TENSOR_COPY_SUCCESSFULLY(GPU, GPU);
/****************************TENSOR_TEST_COPY****************************** */



/***************************TENSOR_TEST_MOVE******************************* */
#define TENSOR_MOVE_TEST_CLASS(device_from, device_to) \
  class TensorMoveTest##device_from##2##device_to : public ::testing::Test { \
   protected: \
    void SetUp() override { \
      another = \
        std::move(std::make_unique<my_tensor::Tensor>(shape, my_tensor::DeviceType::device_from)); \
      auto func = [](int x) -> float { return x; }; \
      float *data = another->GetMutableData(); \
      SET_DATA_ON_##device_from(data, 6, func); \
      temp_data = another->GetData(); \
      tensor = \
        std::move(std::make_unique<my_tensor::Tensor>(other_shape, my_tensor::DeviceType::device_to)); \
      *tensor = std::move(*another); \
    } \
    std::vector<int> shape {1, 2, 3}; \
    std::vector<int> other_shape {2, 3, 4}; \
    std::unique_ptr<my_tensor::Tensor> another; \
    std::unique_ptr<my_tensor::Tensor> tensor; \
    const float *temp_data; \
  };

TENSOR_MOVE_TEST_CLASS(CPU, CPU)
TENSOR_MOVE_TEST_CLASS(CPU, GPU)
TENSOR_MOVE_TEST_CLASS(GPU, CPU)
TENSOR_MOVE_TEST_CLASS(GPU, GPU)

TENSOR_MOVE_OR_COPY_BASIC_TEST(TensorMoveTest)

#define TENSOR_MOVE_SUCCESSFULLY(device_from, device_to) \
TEST_F(TensorMoveTest##device_from##2##device_to, data_move) { \
  EXPECT_EQ(tensor->GetData(), temp_data); \
  EXPECT_EQ(another->GetData(), nullptr); \
}

TENSOR_MOVE_SUCCESSFULLY(CPU, CPU)
TENSOR_MOVE_SUCCESSFULLY(CPU, GPU)
TENSOR_MOVE_SUCCESSFULLY(GPU, CPU)
TENSOR_MOVE_SUCCESSFULLY(GPU, GPU)
/***************************TENSOR_TEST_MOVE******************************* */



/****************************TENSOR_TEST_CPU******************************* */
#define SET_UP_SIX_ELEMENTS_FOR_TEST_CPU(device) \
  std::vector<int> shape {1, 2, 3}; \
  TENSOR_CONSTRUCT_ON_##device(shape, tensor); \
  float* data = tensor->GetMutableData(); \
  auto func = [](int x) -> float { return static_cast<float>(x); }; \
  SET_DATA_ON_##device(data, 6, func); \
  auto another = std::make_unique<my_tensor::Tensor>(tensor->cpu()); \

TEST(tensor_test_cpu, tensor_test_cpu_on_cpu) {
  SET_UP_SIX_ELEMENTS_FOR_TEST_CPU(CPU);
  TENSOR_EXPECT_SHAPE(another, shape);
  TENSOR_EXPECT_ON_CPU(another);
}

TEST(tensor_test_cpu, tensor_test_cpu_on_gpu) {
  SET_UP_SIX_ELEMENTS_FOR_TEST_CPU(GPU);
  TENSOR_EXPECT_SHAPE(another, shape);
  TENSOR_EXPECT_ON_CPU(another);
}

TEST(tensor_test_cpu, tensor_test_cpu_data_on_cpu) {
  SET_UP_SIX_ELEMENTS_FOR_TEST_CPU(CPU);
  TENSOR_DATA_ON_CPU(another);
  TENSOR_EXPECT_EQ_DATA_CPU_CPU(another, tensor);
}

TEST(tensor_test_cpu, tensor_test_cpu_data_on_gpu) {
  SET_UP_SIX_ELEMENTS_FOR_TEST_CPU(GPU);
  TENSOR_DATA_ON_CPU(another);
  TENSOR_EXPECT_EQ_DATA_CPU_GPU(another, tensor);
}
/****************************TENSOR_TEST_CPU******************************* */


/****************************TENSOR_TEST_GPU******************************* */
#define SET_UP_SIX_ELEMENTS_FOR_TEST_GPU(device) \
  std::vector<int> shape {1, 2, 3}; \
  TENSOR_CONSTRUCT_ON_##device(shape, tensor); \
  float* data = tensor->GetMutableData(); \
  auto func = [](int x) -> float { return static_cast<float>(x); }; \
  SET_DATA_ON_##device(data, 6, func); \
  auto another = std::make_unique<my_tensor::Tensor>(tensor->gpu());

TEST(tensor_test_gpu, tensor_test_gpu_on_cpu) {
  SET_UP_SIX_ELEMENTS_FOR_TEST_GPU(CPU);
  TENSOR_EXPECT_SHAPE(another, shape);
  TENSOR_EXPECT_ON_GPU(another);
}

TEST(tensor_test_gpu, tensor_test_gpu_on_gpu) {
  SET_UP_SIX_ELEMENTS_FOR_TEST_GPU(GPU);
  TENSOR_EXPECT_SHAPE(another, shape);
  TENSOR_EXPECT_ON_GPU(another);
}

TEST(tensor_test_gpu, tensor_test_gpu_data_on_cpu) {
  SET_UP_SIX_ELEMENTS_FOR_TEST_GPU(CPU);
  TENSOR_DATA_ON_GPU(another);
  TENSOR_EXPECT_EQ_DATA_CPU_GPU(tensor, another);
}

TEST(tensor_test_gpu, tensor_test_gpu_data_on_gpu) {
  SET_UP_SIX_ELEMENTS_FOR_TEST_GPU(GPU);
  TENSOR_DATA_ON_GPU(another);
  TENSOR_EXPECT_EQ_DATA_GPU_GPU(tensor, another);
}
/****************************TENSOR_TEST_GPU******************************* */



int main(int argc, char** argv) {
  ::testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}
